/**
 Cで学ぶアルゴリズムとデータ構造
 ステップバイステップでＮ−クイーン問題を最適化
 一般社団法人  共同通信社  情報技術局  鈴木  維一郎(suzuki.iichiro@kyodonews.jp)

 必要なこと
　１．$ lspci | grep -i  nvidia 
      で、nVidiaが存在しなければ絶対動かない。
　２．$ nvidia-smi
　　　で、以下の通りに出力され、ＧＰＵの存在が確認できなければ絶対に動かない。

bash-4.2$ nvidia-smi
Wed Jun 27 02:36:34 2018       
+-----------------------------------------------------------------------------+
| NVIDIA-SMI 384.111                Driver Version: 384.111                   |
|-------------------------------+----------------------+----------------------+
| GPU  Name        Persistence-M| Bus-Id        Disp.A | Volatile Uncorr. ECC |
| Fan  Temp  Perf  Pwr:Usage/Cap|         Memory-Usage | GPU-Util  Compute M. |
|===============================+======================+======================|
|   0  Tesla K80           On   | 00000000:00:1E.0 Off |                    0 |
| N/A   38C    P8    30W / 149W |      1MiB / 11439MiB |      0%      Default |
+-------------------------------+----------------------+----------------------+
                                                                               
+-----------------------------------------------------------------------------+
| Processes:                                                       GPU Memory |
|  GPU       PID   Type   Process name                             Usage      |
|=============================================================================|
|  No running processes found                                                 |
+-----------------------------------------------------------------------------+


　３．nVidia CUDAのインストールは思った以上に難しい。
　　　特にMacはOSのバージョン、xcode、command line toolsとの相性もある。
     
  手軽なのは、有料だが Amazon AWSのEC2でCUDA nVidia GPU 対応のサーバーを使うこと。
　（こちらが絶対的におすすめ）


  ４．ここまでが了解であれば、以下のコマンドで実行可能だ。


 コンパイルと実行

 # CPUだけの実行
 $ nvcc gpuNQueen.cu -o gpuNQueen && ./gpuNQueen -cpu 

 # GPUだけの実行
 $ nvcc gpuNQueen.cu -o gpuNQueen && ./gpuNQueen -gpu

 # CPUとGPUの実行
 $ nvcc gpuNQueen.cu -o gpuNQueen && ./gpuNQueen



 １３．ＧＰＵ nVidia-CUDA               N17=    1.67
 *
 *  実行結果
CPU
 N:          Total        Unique                 dd:hh:mm:ss.ms
 4:                 2                 0          00:00:00:00.00
 5:                10                 0          00:00:00:00.00
 6:                 4                 0          00:00:00:00.00
 7:                40                 0          00:00:00:00.00
 8:                92                 0          00:00:00:00.00
 9:               352                 0          00:00:00:00.00
10:               724                 0          00:00:00:00.00
11:              2680                 0          00:00:00:00.00
12:             14200                 0          00:00:00:00.00
13:             73712                 0          00:00:00:00.05
14:            365596                 0          00:00:00:00.29
15:           2279184                 0          00:00:00:01.94

GPU
 N:          Total        Unique                 dd:hh:mm:ss.ms
 4:                 2                 0          00:00:00:00.06
 5:                10                 0          00:00:00:00.00
 6:                 4                 0          00:00:00:00.00
 7:                40                 0          00:00:00:00.00
 8:                92                 0          00:00:00:00.00
 9:               352                 0          00:00:00:00.00
10:               724                 0          00:00:00:00.00
11:              2680                 0          00:00:00:00.00
12:             14200                 0          00:00:00:00.01
13:             73712                 0          00:00:00:00.02
14:            365596                 0          00:00:00:00.01
15:           2279184                 0          00:00:00:00.05
16:          14772512                 0          00:00:00:00.27
17:          95815104                 0          00:00:00:01.65

*/
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define THREAD_NUM		96
#define MAX 15

long Total=0 ;        //合計解
long Unique=0;
int down[2*MAX-1]; //down:flagA 縦 配置フラグ　
int left[2*MAX-1];  //left:flagB 斜め配置フラグ　
int right[2*MAX-1];  //right:flagC 斜め配置フラグ　
int aBoard[2*MAX-1];      //aBoard[] チェス盤の横一列

/**
  case 1 : 再帰　非CUDA 07_04相当
 07_04
 15:      2279184               0           13.50
  1. バックトラック

 N:          Total        Unique                 dd:hh:mm:ss.ms
 4:                 2                 0          00:00:00:00.00
 5:                10                 0          00:00:00:00.00
 6:                 4                 0          00:00:00:00.00
 7:                40                 0          00:00:00:00.00
 8:                92                 0          00:00:00:00.00
 9:               352                 0          00:00:00:00.00
10:               724                 0          00:00:00:00.00
11:              2680                 0          00:00:00:00.01
12:             14200                 0          00:00:00:00.08
13:             73712                 0          00:00:00:00.45
14:            365596                 0          00:00:00:02.72
15:           2279184                 0          00:00:00:17.61
**/
void solve_nqueen_Recursive_BT(int row,int size){
  if(row==size){
    Total++;
  }else{
    for(int i=0;i<size;i++){
      aBoard[row]=i ;
      if(down[i]==0&&left[row-i+(size-1)]==0&&right[row+i]==0){
        down[i]=left[row-aBoard[row]+size-1]=right[row+aBoard[row]]=1; 
        solve_nqueen_Recursive_BT(row+1,size);
        down[i]=left[row-aBoard[row]+size-1]=right[row+aBoard[row]]=0; 
      }
    }  
  }
}

/**
  case 2 : 非再帰　非CUDA
  1. バックトラック

 N:          Total        Unique                 dd:hh:mm:ss.ms
 4:                 2                 0          00:00:00:00.00
 5:                10                 0          00:00:00:00.00
 6:                 4                 0          00:00:00:00.00
 7:                40                 0          00:00:00:00.00
 8:                92                 0          00:00:00:00.00
 9:               352                 0          00:00:00:00.00
10:               724                 0          00:00:00:00.00
11:              2680                 0          00:00:00:00.01
12:             14200                 0          00:00:00:00.07
13:             73712                 0          00:00:00:00.44
14:            365596                 0          00:00:00:02.72
15:           2279184                 0          00:00:00:17.41
**/
void solve_nqueen_nonRecursive_BT(int r,int n){
  bool matched;
  while(r>=0) {
    matched=false;
    for(int i=aBoard[r]+1;i<n;i++) {
      if(0==down[i] && 0==left[r+(n-1)-i] && 0==right[r+i]) {
        if(aBoard[r] >= 0) {
          down[aBoard[r]]=left[r+(n-1)-aBoard[r]]=right[r+aBoard[r]]=0;
        }
        aBoard[r]=i;
        down[i]=left[r+(n-1)-i]=right[r+i]=1;
        matched=true;
        break;
      }
    }
    if(matched){
      r++;
      if(r==n){
        Total++;
        r--;
      }
    }else{
      if(aBoard[r]>=0){
        int tmp=aBoard[r];
        aBoard[r]=-1;
        down[tmp]=left[r+(n-1)-tmp]=right[r+tmp]=0;
      }
      r--;
    }
  }
}
/** 
  case 3 : 再帰 非CUDA
  1. バックトラック
  2. ビットマップ

 N:          Total        Unique                 dd:hh:mm:ss.ms
 4:                 2                 0          00:00:00:00.00
 5:                10                 0          00:00:00:00.00
 6:                 4                 0          00:00:00:00.00
 7:                40                 0          00:00:00:00.00
 8:                92                 0          00:00:00:00.00
 9:               352                 0          00:00:00:00.00
10:               724                 0          00:00:00:00.00
11:              2680                 0          00:00:00:00.00
12:             14200                 0          00:00:00:00.01
13:             73712                 0          00:00:00:00.07
14:            365596                 0          00:00:00:00.45
15:           2279184                 0          00:00:00:02.81
*/
long long solve_nqueen_Recursive_BT_BM(int size,unsigned int left,unsigned int down,unsigned int right) {
  unsigned int mask=(1<<size)-1;
  if(down==mask){
    return 1;
  }
	unsigned int bitmap=(left|down|right);
	if((bitmap&mask)==mask){
    return 0;
  }
	long long total=0;
	unsigned int bit=(bitmap+1)&~bitmap;
	while(bit&mask){
		total+=solve_nqueen_Recursive_BT_BM(size,(left|bit)<<1,down|bit,(right|bit)>>1);
		bitmap|=bit;
		bit=(bitmap+1)&~bitmap;
	}
	return total;
}

/** 
  case 4 : 非再帰 非CUDA
  1. バックトラック
  2. ビットマップ

 N:          Total        Unique                 dd:hh:mm:ss.ms
 4:                 2                 0          00:00:00:00.00
 5:                10                 0          00:00:00:00.00
 6:                 4                 0          00:00:00:00.00
 7:                40                 0          00:00:00:00.00
 8:                92                 0          00:00:00:00.00
 9:               352                 0          00:00:00:00.00
10:               724                 0          00:00:00:00.00
11:              2680                 0          00:00:00:00.00
12:             14200                 0          00:00:00:00.00
13:             73712                 0          00:00:00:00.04
14:            365596                 0          00:00:00:00.22
15:           2279184                 0          00:00:00:01.49
*/
long long solve_nqueen_nonRecursive_BT_BM(int n){
  unsigned int down[32];unsigned int left[32];unsigned int right[32];unsigned int bm[32];
  if(n<=0||n>32){return 0;}
  const unsigned int msk=(1<<n)-1;long long total=0;long long uTotal=0;
  int i=0;int j=0;unsigned int bit;
  down[0]=0;left[0]=0;right[0]=0;bm[0]=0;
  for(j=0;j<(n+1)/2;j++){
    bit=(1<<j);
    bm[0]|=bit;down[1]=bit;left[1]=bit<<1;right[1]=bit>>1;
    bm[1]=(down[1]|left[1]|right[1]);
    i=1;
    if(n%2==1&&j==(n+1)/2-1){uTotal=total;total=0;}
    while(i>0){
      if((bm[i]&msk)==msk){i--;}
      else{
        bit=((bm[i]+1)^bm[i])&~bm[i];
        bm[i]|=bit;
        if((bit&msk)!=0){
          if(i+1==n){total++;i--;}
          else{
            down[i+1]=down[i]|bit;left[i+1]=(left[i]|bit)<<1;right[i+1]=(right[i]|bit)>>1;
            bm[i+1]=(down[i+1]|left[i+1]|right[i+1]);
            i++;
          }
        }else{i--;}
      }
    }
  }
  if(n%2==0){return total*2;}
  else{return uTotal*2+total;}
}

/**
  case 5 : 再帰 非CUDA 07_08相当
  07_08
  15:      2279184          285053            5.88

  1. バックトラック BT
  2. ビットマップ   BM
  3. 対象解除法     SO

5. 再帰＋バックトラック(BT)＋ビットマップ(BM)＋対象解除法(SO)
 N:          Total        Unique                 dd:hh:mm:ss.ms
 4:                 2                 1          00:00:00:00.00
 5:                10                 2          00:00:00:00.00
 6:                 4                 1          00:00:00:00.00
 7:                40                 6          00:00:00:00.00
 8:                92                12          00:00:00:00.00
 9:               352                46          00:00:00:00.00
10:               724                92          00:00:00:00.00
11:              2680               341          00:00:00:00.00
12:             14200              1787          00:00:00:00.02
13:             73712              9233          00:00:00:00.14
14:            365596             45752          00:00:00:00.83
15:           2279184            285053          00:00:00:05.61
*/
int aT[MAX];
int aS[MAX];
int bit;
int C2=0;
int C4=0;
int C8=0;

long getTotal();
long getUnique();
void rotate_bitmap(int bf[],int af[],int si);
void vMirror_bitmap(int bf[],int af[],int si);
int rh(int a,int sz);
int intncmp(int lt[],int rt[],int si);
void symmetryOps_bm(int si);

void solve_nqueen_Recursive_BT_BM_SO(int size,int mask,int row,int left,int down,int right){
  int bitmap=mask&~(left|down|right); //配置可能フィールド
  if(row==size){
    if(!bitmap){
      aBoard[row]=bitmap;
      symmetryOps_bm(size);
    }
  }else{
    while(bitmap) {
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap); //最も下位の１ビットを抽出
      solve_nqueen_Recursive_BT_BM_SO(size,mask,row+1,(left|bit)<<1,down|bit,(right|bit)>>1);
    }
  } 
}

long getUnique(){ 
  return C2+C4+C8;
}
long getTotal(){ 
  return C2*2+C4*4+C8*8;
}
void rotate_bitmap(int bf[],int af[],int si){
  for(int i=0;i<si;i++){
    int t=0;
    for(int j=0;j<si;j++){
      t|=((bf[j]>>i)&1)<<(si-j-1); // x[j] の i ビット目を
    }
    af[i]=t;                        // y[i] の j ビット目にする
  }
}
void vMirror_bitmap(int bf[],int af[],int si){
  int score ;
  for(int i=0;i<si;i++) {
    score=bf[i];
    af[i]=rh(score,si-1);
  }
}
int rh(int a,int sz){
  int tmp=0;
  for(int i=0;i<=sz;i++){
    if(a&(1<<i)){ return tmp|=(1<<(sz-i)); }
  }
  return tmp;
}
int intncmp(int lt[],int rt[],int si){
  int rtn=0;
  for(int k=0;k<si;k++){
    rtn=lt[k]-rt[k];
    if(rtn!=0){ break;}
  }
  return rtn;
}
void symmetryOps_bm(int si){
  int nEquiv;
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<si;i++){ aT[i]=aBoard[i];}
  rotate_bitmap(aT,aS,si);    //時計回りに90度回転
  int k=intncmp(aBoard,aS,si);
  if(k>0)return;
  if(k==0){ nEquiv=2;}else{
    rotate_bitmap(aS,aT,si);  //時計回りに180度回転
    k=intncmp(aBoard,aT,si);
    if(k>0)return;
    if(k==0){ nEquiv=4;}else{
      rotate_bitmap(aT,aS,si);//時計回りに270度回転
      k=intncmp(aBoard,aS,si);
      if(k>0){ return;}
      nEquiv=8;
    }
  }
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<si;i++){ aS[i]=aBoard[i];}
  vMirror_bitmap(aS,aT,si);   //垂直反転
  k=intncmp(aBoard,aT,si);
  if(k>0){ return; }
  if(nEquiv>2){               //-90度回転 対角鏡と同等       
    rotate_bitmap(aT,aS,si);
    k=intncmp(aBoard,aS,si);
    if(k>0){return;}
    if(nEquiv>4){             //-180度回転 水平鏡像と同等
      rotate_bitmap(aS,aT,si);
      k=intncmp(aBoard,aT,si);
      if(k>0){ return;}       //-270度回転 反対角鏡と同等
      rotate_bitmap(aT,aS,si);
      k=intncmp(aBoard,aS,si);
      if(k>0){ return;}
    }
  }
  if(nEquiv==2){ C2++; }
  if(nEquiv==4){ C4++; }
  if(nEquiv==8){ C8++; }
}


/**
  case 6 : 非再帰 非CUDA
  1. バックトラック BT
  2. ビットマップ   BM
  3. 対象解除法     SO

6. 非再帰＋バックトラック(BT)＋ビットマップ(BM)＋対象解除法(SO)
 N:          Total        Unique                 dd:hh:mm:ss.ms
 4:                 2                 1          00:00:00:00.00
 5:                10                 2          00:00:00:00.00
 6:                 4                 1          00:00:00:00.00
 7:                40                 6          00:00:00:00.00
 8:                92                12          00:00:00:00.00
 9:               352                46          00:00:00:00.00
10:               724                92          00:00:00:00.00
11:              2680               341          00:00:00:00.00
12:             14200              1787          00:00:00:00.03
13:             73712              9233          00:00:00:00.19
14:            365596             45752          00:00:00:01.14
15:           2279184            285053          00:00:00:07.70
*/
struct HIKISU{
  int Y;
  int I;
  int M;
  int L;
  int D;
  int R;
  int B;
};
struct STACK {
  struct HIKISU param[MAX];
  int current;
};
void solve_nqueen_nonRecursive_BT_BM_SO(int n,int msk,int y,int l,int d,int r){
  struct STACK stParam;
  for (int m=0;m<n;m++){ 
    stParam.param[m].Y=0;
    stParam.param[m].I=n;
    stParam.param[m].M=0;
    stParam.param[m].L=0;
    stParam.param[m].D=0;
    stParam.param[m].R=0;
    stParam.param[m].B=0;
  }
  stParam.current=0;
  int bend=0;
  int rflg=0;
  int bm;
  while(1){
  if(rflg==0){
   bm=msk&~(l|d|r); //配置可能フィールド
  }
  if(y==n&&!bm&&rflg==0){
    aBoard[y]=bm;
    symmetryOps_bm(n);
  }else{
    while(bm|| rflg==1) {
        if(rflg==0){
      bm^=aBoard[y]=bit=(-bm&bm); //最も下位の１ビットを抽出
          if(stParam.current<MAX){
            stParam.param[stParam.current].Y=y;
            stParam.param[stParam.current].I=n;
            stParam.param[stParam.current].M=msk;
            stParam.param[stParam.current].L=l;
            stParam.param[stParam.current].D=d;
            stParam.param[stParam.current].R=r;
            stParam.param[stParam.current].B=bm;
            (stParam.current)++;
          }
          y=y+1;
          l=(l|bit)<<1;
          d=(d|bit);
          r=(r|bit)>>1;
          bend=1;
          break;
        }
        if(rflg==1){ 
          if(stParam.current>0){
            stParam.current--;
          }
          n=stParam.param[stParam.current].I;
          y=stParam.param[stParam.current].Y;
          msk=stParam.param[stParam.current].M;
          l=stParam.param[stParam.current].L;
          d=stParam.param[stParam.current].D;
          r=stParam.param[stParam.current].R;
          bm=stParam.param[stParam.current].B;
          rflg=0;
        }
    }
      if(bend==1 && rflg==0){
        bend=0;
        continue;
      }
  } 
    if(y==0){
      break;
    }else{
      //goto ret;
      rflg=1;
    }
  }
}

/**
  case 7 : 再帰 非CUDA 07_09に相当
  07_09
  15:      2279184          285053            3.33

  1. バックトラック BT
  2. ビットマップ   BM
  3. 対象解除法     SO
  4. 最上段のクイーンの位置による枝刈り BOUND 

7. 再帰＋バックトラック(BT)＋ビットマップ(BM)＋対象解除法(SO)＋枝刈り(BOUND)
 N:          Total        Unique                 dd:hh:mm:ss.ms
 4:                 2                 1          00:00:00:00.00
 5:                10                 2          00:00:00:00.00
 6:                 4                 1          00:00:00:00.00
 7:                40                 6          00:00:00:00.00
 8:                92                12          00:00:00:00.00
 9:               352                46          00:00:00:00.00
10:               724                92          00:00:00:00.00
11:              2680               341          00:00:00:00.00
12:             14200              1787          00:00:00:00.01
13:             73712              9233          00:00:00:00.08
14:            365596             45752          00:00:00:00.52
15:           2279184            285053          00:00:00:03.42

*/
int BOUND1;
int BOUND2;
int TOPBIT;
int ENDBIT;
int SIDEMASK;
int LASTMASK;

void backTrack2_Recursive_BT_BM_SO_BOUND(int size,int mask,int row,int l,int d,int r){
  int bit;
  int bitmap=mask&~(l|d|r); /* 配置可能フィールド */
  if (row==size) {
    if(!bitmap){
      aBoard[row]=bitmap;
      symmetryOps_bm(size);
    }
  }else{
    while(bitmap) {
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap); //最も下位の１ビットを抽出
      backTrack2_Recursive_BT_BM_SO_BOUND(size,mask,row+1,(l|bit)<<1,d|bit,(r|bit)>>1);
    }
  } 
}
void backTrack1_Recursive_BT_BM_SO_BOUND(int size,int mask,int row,int l,int d,int r){
  int bit;
  int bitmap=mask&~(l|d|r); /* 配置可能フィールド */
  if (row==size) {
    if(!bitmap){
      aBoard[row]=bitmap;
      symmetryOps_bm(size);
    }
  }else{
    while(bitmap) {
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap); //SO最も下位の１ビットを抽出
      backTrack1_Recursive_BT_BM_SO_BOUND(size,mask,row+1,(l|bit)<<1,d|bit,(r|bit)>>1);
    }
  } 
}
void solve_nqueen_Recursive_BT_BM_SO_BOUND(int size,int mask) {
  int bit;
  TOPBIT=1<<(size-1);
  aBoard[0]=1;
  for(BOUND1=2;BOUND1<size-1;BOUND1++){
    aBoard[1]=bit=(1<<BOUND1);
    backTrack1_Recursive_BT_BM_SO_BOUND(size,mask,2,(2|bit)<<1,(1|bit),(bit>>1));
  }
  SIDEMASK=LASTMASK=(TOPBIT|1);
  ENDBIT=(TOPBIT>>1);
  for(BOUND1=1,BOUND2=size-2;BOUND1<BOUND2;BOUND1++,BOUND2--){
    aBoard[0]=bit=(1<<BOUND1);
    backTrack2_Recursive_BT_BM_SO_BOUND(size,mask,1,bit<<1,bit,bit>>1);
    LASTMASK|=LASTMASK>>1|LASTMASK<<1;
    ENDBIT>>=1;
  }
}

/**
  case 8 : 非再帰 非CUDA
  1. バックトラック BT
  2. ビットマップ   BM
  3. 対象解除法     SO
  4. 最上段のクイーンの位置による枝刈り BOUND

 N:          Total        Unique                 dd:hh:mm:ss.ms
 4:                 2                 1          00:00:00:00.00
 5:                10                 2          00:00:00:00.00
 6:                 4                 1          00:00:00:00.00
 7:                40                 6          00:00:00:00.00
 8:                92                12          00:00:00:00.00
 9:               352                46          00:00:00:00.00
10:               724                92          00:00:00:00.00
11:              2680               341          00:00:00:00.00
12:             14200              1787          00:00:00:00.02
13:             73712              9233          00:00:00:00.10
14:            365596             45752          00:00:00:00.67
15:           2279184            285053          00:00:00:04.22
*/
//long long solve_nqueen_nonRecursive_BT_BM_SO_BOUND(int n){
//  return true;
//}
void backTrack2_nonRecursive_BT_BM_SO_BOUND(int is,int msk,int y, int l, int d, int r);
void backTrack1_nonRecursive_BT_BM_SO_BOUND(int si,int msk,int y, int l, int d, int r);
void solve_nqueen_nonRecursive_BT_BM_SO_BOUND(int n,int B1,int B2,int msk){
  int bit;
  if(B1==0){
    aBoard[0]=1;
    for(BOUND1=2;BOUND1<n-1;BOUND1++){
      aBoard[1]=bit=(1<<BOUND1);
      backTrack1_nonRecursive_BT_BM_SO_BOUND(n,msk,2,(2|bit)<<1,(1|bit),(bit>>1));
    }
  } else{
    BOUND1=B1;
    BOUND2=B2;
    if(BOUND1<BOUND2){
      aBoard[0]=bit=(1<<BOUND1);
      backTrack2_nonRecursive_BT_BM_SO_BOUND(n,msk,1,bit<<1,bit,bit>>1);
    }
  }
}
void backTrack2_nonRecursive_BT_BM_SO_BOUND(int si,int msk,int y,int l,int d,int r){
  struct STACK stParam_2;
  for (int m=0;m<si;m++){ 
    stParam_2.param[m].Y=0;
    stParam_2.param[m].I=si;
    stParam_2.param[m].M=0;
    stParam_2.param[m].L=0;
    stParam_2.param[m].D=0;
    stParam_2.param[m].R=0;
    stParam_2.param[m].B=0;
  }
  stParam_2.current=0;
  int bend_2=0;
  int rflg_2=0;
  int bit;
  int bm;
  while(1){
//start:
    if(rflg_2==0){
      bm=msk&~(l|d|r); /* 配置可能フィールド */
    }
    if (y==si&&rflg_2==0) {
      if(!bm){
        aBoard[y]=bm;
        symmetryOps_bm(si);
      }
    }else{
      while(bm|| rflg_2==1) {
        if(rflg_2==0){
          bm^=aBoard[y]=bit=(-bm&bm); //最も下位の１ビットを抽出
          if(stParam_2.current<MAX){
            stParam_2.param[stParam_2.current].Y=y;
            stParam_2.param[stParam_2.current].I=si;
            stParam_2.param[stParam_2.current].M=msk;
            stParam_2.param[stParam_2.current].L=l;
            stParam_2.param[stParam_2.current].D=d;
            stParam_2.param[stParam_2.current].R=r;
            stParam_2.param[stParam_2.current].B=bm;
            (stParam_2.current)++;
          }
          y=y+1;
          l=(l|bit)<<1;
          d=(d|bit);
          r=(r|bit)>>1;
          bend_2=1;
          break;
        }
        if(rflg_2==1){ 
          if(stParam_2.current>0){
            stParam_2.current--;
          }
          si=stParam_2.param[stParam_2.current].I;
          y=stParam_2.param[stParam_2.current].Y;
          msk=stParam_2.param[stParam_2.current].M;
          l=stParam_2.param[stParam_2.current].L;
          d=stParam_2.param[stParam_2.current].D;
          r=stParam_2.param[stParam_2.current].R;
          bm=stParam_2.param[stParam_2.current].B;
          rflg_2=0;
        }
      }
      if(bend_2==1 && rflg_2==0){
        bend_2=0;
        continue;
      }
    } 
    if(y==1){
      break;
    }else{
      rflg_2=1;
    }
  }
}
void backTrack1_nonRecursive_BT_BM_SO_BOUND(int si,int msk,int y,int l,int d,int r){
  struct STACK stParam_1;
  for (int m=0;m<si;m++){ 
    stParam_1.param[m].Y=0;
    stParam_1.param[m].I=si;
    stParam_1.param[m].M=0;
    stParam_1.param[m].L=0;
    stParam_1.param[m].D=0;
    stParam_1.param[m].R=0;
    stParam_1.param[m].B=0;
  }
  stParam_1.current=0;
  int bend_1=0;
  int rflg_1=0;
  int bit;
  int bm;
  while(1){
    if(rflg_1==0){
      bm=msk&~(l|d|r); /* 配置可能フィールド */
    }
    if (y==si&&rflg_1==0) {
      if(!bm){
        aBoard[y]=bm;
        symmetryOps_bm(si);
      }
    }else{
      while(bm|| rflg_1==1) {
        if(rflg_1==0){
          bm^=aBoard[y]=bit=(-bm&bm); //最も下位の１ビットを抽出
          if(stParam_1.current<MAX){
            stParam_1.param[stParam_1.current].Y=y;
            stParam_1.param[stParam_1.current].I=si;
            stParam_1.param[stParam_1.current].M=msk;
            stParam_1.param[stParam_1.current].L=l;
            stParam_1.param[stParam_1.current].D=d;
            stParam_1.param[stParam_1.current].R=r;
          stParam_1.param[stParam_1.current].B=bm;
            (stParam_1.current)++;
          }
          y=y+1;
          l=(l|bit)<<1;
          d=(d|bit);
          r=(r|bit)>>1;
          bend_1=1;
          break;
        }
//ret:
        if(rflg_1==1){ 
        if(stParam_1.current>0){
          stParam_1.current--;
        }
        si=stParam_1.param[stParam_1.current].I;
        y=stParam_1.param[stParam_1.current].Y;
        msk=stParam_1.param[stParam_1.current].M;
        l=stParam_1.param[stParam_1.current].L;
        d=stParam_1.param[stParam_1.current].D;
        r=stParam_1.param[stParam_1.current].R;
        bm=stParam_1.param[stParam_1.current].B;
          rflg_1=0;
        }
      }
      if(bend_1==1 && rflg_1==0){
        bend_1=0;
        continue;
      }
    } 
    if(y==2){
      break;
    }else{
      rflg_1=1;
    }
  }
}
/**
  case 9 : 再帰 非CUDA 07_10相当
  07_10
  15:      2279184          285053            1.48

  1. バックトラック BT
  2. ビットマップ   BM
  3. 対象解除法     SO
  4. 最上段のクイーンの位置による枝刈り BOUND 
  5. BOUNDの枝刈り

 N:          Total        Unique                 dd:hh:mm:ss.ms
 4:                 2                 1          00:00:00:00.00
 5:                10                 2          00:00:00:00.00
 6:                 4                 1          00:00:00:00.00
 7:                40                 6          00:00:00:00.00
 8:                92                12          00:00:00:00.00
 9:               352                46          00:00:00:00.00
10:               724                92          00:00:00:00.00
11:              2680               341          00:00:00:00.00
12:             14200              1787          00:00:00:00.00
13:             73712              9233          00:00:00:00.04
14:            365596             45752          00:00:00:00.22
15:           2279184            285053          00:00:00:01.52
*/
void backTrack2_Recursive_BT_BM_SO_BOUND_BOUND2(int size,int mask,int row,int l,int d,int r){
  int bit;
  int bitmap=mask&~(l|d|r); /* 配置可能フィールド */
  //【枝刈り】１行目角にクイーンがある場合回転対称チェックを省略
  //if (row==size) {
  if (row==size-1) {
    //if(!bitmap){
    if(bitmap){
      if((bitmap&LASTMASK)==0){
        aBoard[row]=bitmap;
        symmetryOps_bm(size);
      }
    }
  }else{
    // 追加はじめ
    if(row<BOUND1){             	//【枝刈り】上部サイド枝刈り
      bitmap&=~SIDEMASK;
    }else if(row==BOUND2) {     	//【枝刈り】下部サイド枝刈り
      if((d&SIDEMASK)==0){ return; }
      if((d&SIDEMASK)!=SIDEMASK){ bitmap&=SIDEMASK; }
    }
    // 追加終わり
    while(bitmap) {
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap); //最も下位の１ビットを抽出
      backTrack2_Recursive_BT_BM_SO_BOUND_BOUND2(size,mask,row+1,(l|bit)<<1,d|bit,(r|bit)>>1);
    }
  } 
}
void backTrack1_Recursive_BT_BM_SO_BOUND_BOUND2(int size,int mask,int row,int l,int d,int r){
  int bit;
  int bitmap=mask&~(l|d|r); /* 配置可能フィールド */
  //【枝刈り】１行目角にクイーンがある場合回転対称チェックを省略
  //if (row==size) {
  if (row==size-1) {
    //if(!bitmap){
    if(bitmap){
      aBoard[row]=bitmap;
      //symmetryOps_bm(size);
      C8++;
    }
  }else{
		//【枝刈り】鏡像についても主対角線鏡像のみを判定すればよい
		// ２行目、２列目を数値とみなし、２行目＜２列目という条件を課せばよい
    if(row<BOUND1) {
      bitmap&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
    }
    while(bitmap) {
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap); //SO最も下位の１ビットを抽出
      backTrack1_Recursive_BT_BM_SO_BOUND_BOUND2(size,mask,row+1,(l|bit)<<1,d|bit,(r|bit)>>1);
    }
  } 
}
void solve_nqueen_Recursive_BT_BM_SO_BOUND_BOUND2(int size,int mask) {
  int bit;
  TOPBIT=1<<(size-1);
  aBoard[0]=1;
  for(BOUND1=2;BOUND1<size-1;BOUND1++){
    aBoard[1]=bit=(1<<BOUND1);
    backTrack1_Recursive_BT_BM_SO_BOUND_BOUND2(size,mask,2,(2|bit)<<1,(1|bit),(bit>>1));
  }
  SIDEMASK=LASTMASK=(TOPBIT|1);
  ENDBIT=(TOPBIT>>1);
  for(BOUND1=1,BOUND2=size-2;BOUND1<BOUND2;BOUND1++,BOUND2--){
    aBoard[0]=bit=(1<<BOUND1);
    backTrack2_Recursive_BT_BM_SO_BOUND_BOUND2(size,mask,1,bit<<1,bit,bit>>1);
    LASTMASK|=LASTMASK>>1|LASTMASK<<1;
    ENDBIT>>=1;
  }
}




/**
  case 10 : 非再帰 非CUDA
  1. バックトラック BT
  2. ビットマップ   BM
  3. 対象解除法     SO
  4. 最上段のクイーンの位置による枝刈り BOUND 
  5. BOUNDの枝刈り

 N:          Total        Unique                 dd:hh:mm:ss.ms
 4:                 2                 1          00:00:00:00.00
 5:                10                 2          00:00:00:00.00
 6:                 4                 1          00:00:00:00.00
 7:                40                 6          00:00:00:00.00
 8:                92                12          00:00:00:00.00
 9:               352                46          00:00:00:00.00
10:               724                92          00:00:00:00.00
11:              2680               341          00:00:00:00.00
12:             14200              1787          00:00:00:00.01
13:             73712              9233          00:00:00:00.05
14:            365596             45752          00:00:00:00.29
15:           2279184            285053          00:00:00:01.94
*/

void backTrack2_nonRecursive_BT_BM_SO_BOUND_BOUND2(int size,int mask,int row, int l, int d, int r){
  struct STACK stParam_2;
  for (int m=0;m<size;m++){ 
    stParam_2.param[m].Y=0;
    stParam_2.param[m].I=size;
    stParam_2.param[m].M=0;
    stParam_2.param[m].L=0;
    stParam_2.param[m].D=0;
    stParam_2.param[m].R=0;
    stParam_2.param[m].B=0;
  }
  stParam_2.current=0;
  int bend_2=0;
  int rflg_2=0;
  int bitmap;
  int bit;
  while(1){
   if(rflg_2==0){ 
    bitmap=mask&~(l|d|r); /* 配置可能フィールド */
   }
  //【枝刈り】１行目角にクイーンがある場合回転対称チェックを省略
    if (row==size-1&&rflg_2==0) {
      if(bitmap){
        if((bitmap&LASTMASK)==0){
          aBoard[row]=bitmap;
          symmetryOps_bm(size);
        }
      }
    }else{
    // 追加はじめ
        if(row<BOUND1&&rflg_2==0){             	//【枝刈り】上部サイド枝刈り
          bitmap&=~SIDEMASK;
        }else if(row==BOUND2&&rflg_2==0) {     	//【枝刈り】下部サイド枝刈り
          if((d&SIDEMASK)==0&&rflg_2==0){ 
            rflg_2=1;
          }
          if((d&SIDEMASK)!=SIDEMASK&&rflg_2==0){ bitmap&=SIDEMASK; }
        }
    // 追加終わり
      while(bitmap||rflg_2==1) {
        if(rflg_2==0){
          bitmap^=aBoard[row]=bit=(-bitmap&bitmap); //最も下位の１ビットを抽出
          //backTrack2_nonRecursive_BT_BM_SO_BOUND_BOUND2(size,mask,row+1,(l|bit)<<1,d|bit,(r|bit)>>1);
          if(stParam_2.current<MAX){
            stParam_2.param[stParam_2.current].Y=row;
            stParam_2.param[stParam_2.current].I=size;
            stParam_2.param[stParam_2.current].M=mask;
            stParam_2.param[stParam_2.current].L=l;
            stParam_2.param[stParam_2.current].D=d;
            stParam_2.param[stParam_2.current].R=r;
            stParam_2.param[stParam_2.current].B=bitmap;
            (stParam_2.current)++;
          }
          row=row+1;
          l=(l|bit)<<1;
          d=(d|bit);
          r=(r|bit)>>1;
          bend_2=1;
          break;
        }
        if(rflg_2==1){ 
          if(stParam_2.current>0){
            stParam_2.current--;
          }
          size=stParam_2.param[stParam_2.current].I;
          row=stParam_2.param[stParam_2.current].Y;
          mask=stParam_2.param[stParam_2.current].M;
          l=stParam_2.param[stParam_2.current].L;
          d=stParam_2.param[stParam_2.current].D;
          r=stParam_2.param[stParam_2.current].R;
          bitmap=stParam_2.param[stParam_2.current].B;
          rflg_2=0;
        }
      }
      if(bend_2==1 && rflg_2==0){
        bend_2=0;
        continue;
      }
    }
    if(row==1){
      break;
    }else{
      rflg_2=1;
    }
  } 
}
void backTrack1_nonRecursive_BT_BM_SO_BOUND_BOUND2(int size,int mask,int row, int l, int d, int r){
  struct STACK stParam_1;
  for (int m=0;m<size;m++){ 
    stParam_1.param[m].Y=0;
    stParam_1.param[m].I=size;
    stParam_1.param[m].M=0;
    stParam_1.param[m].L=0;
    stParam_1.param[m].D=0;
    stParam_1.param[m].R=0;
    stParam_1.param[m].B=0;
  }
  stParam_1.current=0;
  int bend_1=0;
  int rflg_1=0;
  int bit;
  int bitmap;
  while(1){
    if(rflg_1==0){
      bitmap=mask&~(l|d|r); /* 配置可能フィールド */
    }
  //【枝刈り】１行目角にクイーンがある場合回転対称チェックを省略
    if (row==size-1&&rflg_1==0) {
      if(bitmap){
        aBoard[row]=bitmap;
        C8++;
      }
    }else{
		//【枝刈り】鏡像についても主対角線鏡像のみを判定すればよい
		// ２行目、２列目を数値とみなし、２行目＜２列目という条件を課せばよい
      if(row<BOUND1 && rflg_1==0) {
        bitmap&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
      }
      while(bitmap||rflg_1==1) {
        if(rflg_1==0){
          bitmap^=aBoard[row]=bit=(-bitmap&bitmap); //SO最も下位の１ビットを抽出
          //backTrack1_Recursive_BT_BM_SO_BOUND_BOUND2(size,mask,row+1,(l|bit)<<1,d|bit,(r|bit)>>1);
          if(stParam_1.current<MAX){
            stParam_1.param[stParam_1.current].Y=row;
            stParam_1.param[stParam_1.current].I=size;
            stParam_1.param[stParam_1.current].M=mask;
            stParam_1.param[stParam_1.current].L=l;
            stParam_1.param[stParam_1.current].D=d;
            stParam_1.param[stParam_1.current].R=r;
            stParam_1.param[stParam_1.current].B=bitmap;
            (stParam_1.current)++;
          }
          row=row+1;
          l=(l|bit)<<1;
          d=(d|bit);
          r=(r|bit)>>1;
          bend_1=1;
          break;
        }
        if(rflg_1==1){ 
          if(stParam_1.current>0){
            stParam_1.current--;
          }
          size=stParam_1.param[stParam_1.current].I;
          row=stParam_1.param[stParam_1.current].Y;
          mask=stParam_1.param[stParam_1.current].M;
          l=stParam_1.param[stParam_1.current].L;
          d=stParam_1.param[stParam_1.current].D;
          r=stParam_1.param[stParam_1.current].R;
          bitmap=stParam_1.param[stParam_1.current].B;
          rflg_1=0;
        }
      }
      if(bend_1==1 && rflg_1==0){
        bend_1=0;
        continue;
      }
    } 
    if(row==2){
      break;
    }else{
      rflg_1=1;
    }
  } 
}
void solve_nqueen_nonRecursive_BT_BM_SO_BOUND_BOUND2(int size,int mask){
  int bit;
  TOPBIT=1<<(size-1);
  aBoard[0]=1;
  /* 最上段行のクイーンが角にある場合の探索 */
  for(BOUND1=2;BOUND1<size-1;BOUND1++){
    // 角にクイーンを配置 
    aBoard[1]=bit=(1<<BOUND1); 
    //２行目から探索
    backTrack1_nonRecursive_BT_BM_SO_BOUND_BOUND2(size,mask,2,(2|bit)<<1,(1|bit),(bit>>1)); 
  }
  SIDEMASK=LASTMASK=(TOPBIT|1);
  ENDBIT=(TOPBIT>>1);
  /* 最上段行のクイーンが角以外にある場合の探索 
     ユニーク解に対する左右対称解を予め削除するには、
     左半分だけにクイーンを配置するようにすればよい */
  for(BOUND1=1,BOUND2=size-2;BOUND1<BOUND2;BOUND1++,BOUND2--){
    aBoard[0]=bit=(1<<BOUND1);
    backTrack2_nonRecursive_BT_BM_SO_BOUND_BOUND2(size,mask,1,bit<<1,bit,bit>>1);
    LASTMASK|=LASTMASK>>1|LASTMASK<<1;
    ENDBIT>>=1;
  }
}
/**
  case 11 : 再帰 非CUDA 07_11相当
  07_11
  15:      2279184          285053            0.54

  1. バックトラック BT
  2. ビットマップ   BM
  3. 対象解除法     SO
  4. 最上段のクイーンの位置による枝刈り BOUND 
  5. BOUNDの枝刈り
  6. 最適化

 N:          Total        Unique                 dd:hh:mm:ss.ms
 4:                 2                 1          00:00:00:00.00
 5:                10                 2          00:00:00:00.00
 6:                 4                 1          00:00:00:00.00
 7:                40                 6          00:00:00:00.00
 8:                92                12          00:00:00:00.00
 9:               352                46          00:00:00:00.00
10:               724                92          00:00:00:00.00
11:              2680               341          00:00:00:00.00
12:             14200              1787          00:00:00:00.00
13:             73712              9233          00:00:00:00.01
14:            365596             45752          00:00:00:00.08
15:           2279184            285053          00:00:00:00.53
*/
void symmetryOps_Recursive_BT_BM_SO_BOUND_BOUND2_OPT(int size){
  int own,ptn,you,bit;
  //90度回転
  if(aBoard[BOUND2]==1){ own=1; ptn=2;
    while(own<=size-1){ bit=1; you=size-1;
      while((aBoard[you]!=ptn)&&(aBoard[own]>=bit)){ bit<<=1; you--; }
      if(aBoard[own]>bit){ return; } if(aBoard[own]<bit){ break; }
      own++; ptn<<=1;
    }
    /** 90度回転して同型なら180度/270度回転も同型である */
    if(own>size-1){ C2++; return; }
  }
  //180度回転
  if(aBoard[size-1]==ENDBIT){ own=1; you=size-1-1;
    while(own<=size-1){ bit=1; ptn=TOPBIT;
      while((aBoard[you]!=ptn)&&(aBoard[own]>=bit)){ bit<<=1; ptn>>=1; }
      if(aBoard[own]>bit){ return; } if(aBoard[own]<bit){ break; }
      own++; you--;
    }
    /** 90度回転が同型でなくても180度回転が同型である事もある */
    if(own>size-1){ C4++; return; }
  }
  //270度回転
  if(aBoard[BOUND1]==TOPBIT){ own=1; ptn=TOPBIT>>1;
    while(own<=size-1){ bit=1; you=0;
      while((aBoard[you]!=ptn)&&(aBoard[own]>=bit)){ bit<<=1; you++; }
      if(aBoard[own]>bit){ return; } if(aBoard[own]<bit){ break; }
      own++; ptn>>=1;
    }
  }
  C8++;
}
void backTrack2_Recursive_BT_BM_SO_BOUND_BOUND2_OPT(int size,int mask,int row,int left,int down,int right){
	int bit;
	int bitmap=mask&~(left|down|right);
	if(row==size-1){ 								// 【枝刈り】
		if(bitmap){
			if((bitmap&LASTMASK)==0){ 	//【枝刈り】 最下段枝刈り
				aBoard[row]=bitmap;
				symmetryOps_Recursive_BT_BM_SO_BOUND_BOUND2_OPT(size);
			}
		}
	}else{
    if(row<BOUND1){             	//【枝刈り】上部サイド枝刈り
      bitmap&=~SIDEMASK;
    }else if(row==BOUND2) {     	//【枝刈り】下部サイド枝刈り
      if((down&SIDEMASK)==0){ return; }
      if((down&SIDEMASK)!=SIDEMASK){ bitmap&=SIDEMASK; }
    }
		while(bitmap){
			bitmap^=aBoard[row]=bit=(-bitmap&bitmap);
			backTrack2_Recursive_BT_BM_SO_BOUND_BOUND2_OPT(size,mask,row+1,(left|bit)<<1,down|bit,(right|bit)>>1);
		}
	}
}

void backTrack1_Recursive_BT_BM_SO_BOUND_BOUND2_OPT(int size,int mask,int row,int left,int down,int right){
	int bit;
	int bitmap=mask&~(left|down|right);
  //【枝刈り】１行目角にクイーンがある場合回転対称チェックを省略
  if(row==size-1) {
    if(bitmap){
      aBoard[row]=bitmap;
      C8++;
    }
  }else{
		//【枝刈り】鏡像についても主対角線鏡像のみを判定すればよい
		// ２行目、２列目を数値とみなし、２行目＜２列目という条件を課せばよい
    if(row<BOUND1) {
      bitmap&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
    }
		while(bitmap){
			bitmap^=aBoard[row]=bit=(-bitmap&bitmap);
			backTrack1_Recursive_BT_BM_SO_BOUND_BOUND2_OPT(size,mask,row+1,(left|bit)<<1,down|bit,(right|bit)>>1);
		}
	}
}

void solve_nqueen_Recursive_BT_BM_SO_BOUND_BOUND2_OPT(int size,int mask){
	int bit;
	TOPBIT=1<<(size-1);
	aBoard[0]=1;
	for(BOUND1=2;BOUND1<size-1;BOUND1++){
		aBoard[1]=bit=(1<<BOUND1);
		backTrack1_Recursive_BT_BM_SO_BOUND_BOUND2_OPT(size,mask,2,(2|bit)<<1,(1|bit),(bit>>1));
	}
	SIDEMASK=LASTMASK=(TOPBIT|1);
	ENDBIT=(TOPBIT>>1);
	for(BOUND1=1,BOUND2=size-2;BOUND1<BOUND2;BOUND1++,BOUND2--){
		aBoard[0]=bit=(1<<BOUND1);
		backTrack2_Recursive_BT_BM_SO_BOUND_BOUND2_OPT(size,mask,1,bit<<1,bit,bit>>1);
		LASTMASK|=LASTMASK>>1|LASTMASK<<1;
		ENDBIT>>=1;
	}
}


/**
  case 12 : 非再帰 非CUDA
  1. バックトラック BT
  2. ビットマップ   BM
  3. 対象解除法     SO
  4. 最上段のクイーンの位置による枝刈り BOUND 
  5. BOUNDの枝刈り
  6. 最適化

 N:          Total        Unique                 dd:hh:mm:ss.ms
 4:                 2                 1          00:00:00:00.00
 5:                10                 2          00:00:00:00.00
 6:                 4                 1          00:00:00:00.00
 7:                40                 6          00:00:00:00.00
 8:                92                12          00:00:00:00.00
 9:               352                46          00:00:00:00.00
10:               724                92          00:00:00:00.00
11:              2680               341          00:00:00:00.00
12:             14200              1787          00:00:00:00.00
13:             73712              9233          00:00:00:00.02
14:            365596             45752          00:00:00:00.15
15:           2279184            285053          00:00:00:00.97
*/
void backTrack2_nonRecursive_BT_BM_SO_BOUND_BOUND2_OPT(int size,int mask,int row, int l, int d, int r){
  struct STACK stParam_2;
  for (int m=0;m<size;m++){ 
    stParam_2.param[m].Y=0;
    stParam_2.param[m].I=size;
    stParam_2.param[m].M=0;
    stParam_2.param[m].L=0;
    stParam_2.param[m].D=0;
    stParam_2.param[m].R=0;
    stParam_2.param[m].B=0;
  }
  stParam_2.current=0;
  int bend_2=0;
  int rflg_2=0;
  int bitmap;
  int bit;
  while(1){
   if(rflg_2==0){ 
    bitmap=mask&~(l|d|r); /* 配置可能フィールド */
   }
  //【枝刈り】１行目角にクイーンがある場合回転対称チェックを省略
    if (row==size-1&&rflg_2==0) {
      if(bitmap){
        if((bitmap&LASTMASK)==0){
          aBoard[row]=bitmap;
				  symmetryOps_Recursive_BT_BM_SO_BOUND_BOUND2_OPT(size);
        }
      }
    }else{
    // 追加はじめ
        if(row<BOUND1&&rflg_2==0){             	//【枝刈り】上部サイド枝刈り
          bitmap&=~SIDEMASK;
        }else if(row==BOUND2&&rflg_2==0) {     	//【枝刈り】下部サイド枝刈り
          if((d&SIDEMASK)==0&&rflg_2==0){ 
            rflg_2=1;
          }
          if((d&SIDEMASK)!=SIDEMASK&&rflg_2==0){ bitmap&=SIDEMASK; }
        }
    // 追加終わり
      while(bitmap||rflg_2==1) {
        if(rflg_2==0){
          bitmap^=aBoard[row]=bit=(-bitmap&bitmap); //最も下位の１ビットを抽出
          if(stParam_2.current<MAX){
            stParam_2.param[stParam_2.current].Y=row;
            stParam_2.param[stParam_2.current].I=size;
            stParam_2.param[stParam_2.current].M=mask;
            stParam_2.param[stParam_2.current].L=l;
            stParam_2.param[stParam_2.current].D=d;
            stParam_2.param[stParam_2.current].R=r;
            stParam_2.param[stParam_2.current].B=bitmap;
            (stParam_2.current)++;
          }
          row=row+1;
          l=(l|bit)<<1;
          d=(d|bit);
          r=(r|bit)>>1;
          bend_2=1;
          break;
        }
        if(rflg_2==1){ 
          if(stParam_2.current>0){
            stParam_2.current--;
          }
          size=stParam_2.param[stParam_2.current].I;
          row=stParam_2.param[stParam_2.current].Y;
          mask=stParam_2.param[stParam_2.current].M;
          l=stParam_2.param[stParam_2.current].L;
          d=stParam_2.param[stParam_2.current].D;
          r=stParam_2.param[stParam_2.current].R;
          bitmap=stParam_2.param[stParam_2.current].B;
          rflg_2=0;
        }
      }
      if(bend_2==1 && rflg_2==0){
        bend_2=0;
        continue;
      }
    }
    if(row==1){
      break;
    }else{
      rflg_2=1;
    }
  } 
}
void backTrack1_nonRecursive_BT_BM_SO_BOUND_BOUND2_OPT(int size,int mask,int row, int l, int d, int r){
  struct STACK stParam_1;
  for (int m=0;m<size;m++){ 
    stParam_1.param[m].Y=0;
    stParam_1.param[m].I=size;
    stParam_1.param[m].M=0;
    stParam_1.param[m].L=0;
    stParam_1.param[m].D=0;
    stParam_1.param[m].R=0;
    stParam_1.param[m].B=0;
  }
  stParam_1.current=0;
  int bend_1=0;
  int rflg_1=0;
  int bit;
  int bitmap;
  while(1){
    if(rflg_1==0){
      bitmap=mask&~(l|d|r); /* 配置可能フィールド */
    }
  //【枝刈り】１行目角にクイーンがある場合回転対称チェックを省略
    if (row==size-1&&rflg_1==0) {
      if(bitmap){
        aBoard[row]=bitmap;
        C8++;
      }
    }else{
		//【枝刈り】鏡像についても主対角線鏡像のみを判定すればよい
		// ２行目、２列目を数値とみなし、２行目＜２列目という条件を課せばよい
      if(row<BOUND1 && rflg_1==0) {
        bitmap&=~2; // bm|=2; bm^=2; (bm&=~2と同等)
      }
      while(bitmap||rflg_1==1) {
        if(rflg_1==0){
          bitmap^=aBoard[row]=bit=(-bitmap&bitmap); //SO最も下位の１ビットを抽出
          if(stParam_1.current<MAX){
            stParam_1.param[stParam_1.current].Y=row;
            stParam_1.param[stParam_1.current].I=size;
            stParam_1.param[stParam_1.current].M=mask;
            stParam_1.param[stParam_1.current].L=l;
            stParam_1.param[stParam_1.current].D=d;
            stParam_1.param[stParam_1.current].R=r;
            stParam_1.param[stParam_1.current].B=bitmap;
            (stParam_1.current)++;
          }
          row=row+1;
          l=(l|bit)<<1;
          d=(d|bit);
          r=(r|bit)>>1;
          bend_1=1;
          break;
        }
        if(rflg_1==1){ 
          if(stParam_1.current>0){
            stParam_1.current--;
          }
          size=stParam_1.param[stParam_1.current].I;
          row=stParam_1.param[stParam_1.current].Y;
          mask=stParam_1.param[stParam_1.current].M;
          l=stParam_1.param[stParam_1.current].L;
          d=stParam_1.param[stParam_1.current].D;
          r=stParam_1.param[stParam_1.current].R;
          bitmap=stParam_1.param[stParam_1.current].B;
          rflg_1=0;
        }
      }
      if(bend_1==1 && rflg_1==0){
        bend_1=0;
        continue;
      }
    } 
    if(row==2){
      break;
    }else{
      rflg_1=1;
    }
  } 
}
void solve_nqueen_nonRecursive_BT_BM_SO_BOUND_BOUND2_OPT(int size,int mask){
  int bit;
  TOPBIT=1<<(size-1);
  aBoard[0]=1;
  /* 最上段行のクイーンが角にある場合の探索 */
  for(BOUND1=2;BOUND1<size-1;BOUND1++){
    // 角にクイーンを配置 
    aBoard[1]=bit=(1<<BOUND1); 
    //２行目から探索
    backTrack1_nonRecursive_BT_BM_SO_BOUND_BOUND2_OPT(size,mask,2,(2|bit)<<1,(1|bit),(bit>>1)); 
  }
  SIDEMASK=LASTMASK=(TOPBIT|1);
  ENDBIT=(TOPBIT>>1);
  /* 最上段行のクイーンが角以外にある場合の探索 
     ユニーク解に対する左右対称解を予め削除するには、
     左半分だけにクイーンを配置するようにすればよい */
  for(BOUND1=1,BOUND2=size-2;BOUND1<BOUND2;BOUND1++,BOUND2--){
    aBoard[0]=bit=(1<<BOUND1);
    backTrack2_nonRecursive_BT_BM_SO_BOUND_BOUND2_OPT(size,mask,1,bit<<1,bit,bit>>1);
    LASTMASK|=LASTMASK>>1|LASTMASK<<1;
    ENDBIT>>=1;
  }
}
/** #################################################################

  nVidia CUDA ブロック

#####################################################################*/
/** 
  CUDA 非再帰 CPUイテレータから複数の初期条件を受け取り、カウント
  1. バックトラック backTrack
  2. ビットマップ   bitmap
14:            365596                 0          00:00:00:00.08
15:           2279184                 0          00:00:00:00.49
*/
__global__ void solve_nqueen_cuda_kernel_bt_bm(
  int n,int mark,
  unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,
  unsigned int* results,int totalCond){
  const int tid=threadIdx.x,bid=blockIdx.x,idx=bid*blockDim.x+tid;
  __shared__ unsigned int down[THREAD_NUM][10],left[THREAD_NUM][10],right[THREAD_NUM][10],
                          bitmap[THREAD_NUM][10],sum[THREAD_NUM];
  const unsigned int mask=(1<<n)-1;int total=0,i=0;unsigned int bit;
  if(idx<totalCond){
    down[tid][i]=totalDown[idx];
    left[tid][i]=totalLeft[idx];
    right[tid][i]=totalRight[idx];
    bitmap[tid][i]=down[tid][i]|left[tid][i]|right[tid][i];
    while(i>=0){
      if((bitmap[tid][i]&mask)==mask){i--;}
      else{
        bit=(bitmap[tid][i]+1)&~bitmap[tid][i];
        bitmap[tid][i]|=bit;
        if((bit&mask)!=0){
          if(i+1==mark){total++;i--;}
          else{
            down[tid][i+1]=down[tid][i]|bit;
            left[tid][i+1]=(left[tid][i]|bit)<<1;
            right[tid][i+1]=(right[tid][i]|bit)>>1;
            bitmap[tid][i+1]=(down[tid][i+1]|left[tid][i+1]|right[tid][i+1]);
            i++;
          }
        }else{i--;}
      }
    }
    sum[tid]=total;
  }else{sum[tid]=0;} 
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){sum[tid]+=sum[tid+64];} 
  __syncthreads();if(tid<32){sum[tid]+=sum[tid+32];} 
  __syncthreads();if(tid<16){sum[tid]+=sum[tid+16];} 
  __syncthreads();if(tid<8){sum[tid]+=sum[tid+8];} 
  __syncthreads();if(tid<4){sum[tid]+=sum[tid+4];} 
  __syncthreads();if(tid<2){sum[tid]+=sum[tid+2];} 
  __syncthreads();if(tid<1){sum[tid]+=sum[tid+1];} 
  __syncthreads();if(tid==0){results[bid]=sum[0];}
}
long long solve_nqueen_cuda(int n,int steps) {
  unsigned int down[32];unsigned int left[32];unsigned int right[32];
  unsigned int m[32];unsigned int bit;
  if(n<=0||n>32){return 0;}
  unsigned int* totalDown=new unsigned int[steps];
  unsigned int* totalLeft=new unsigned int[steps];
  unsigned int* totalRight=new unsigned int[steps];
  unsigned int* results=new unsigned int[steps];
  unsigned int* downCuda;unsigned int* leftCuda;unsigned int* rightCuda;
  unsigned int* resultsCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  const unsigned int mask=(1<<n)-1;
  const unsigned int mark=n>11?n-10:2;
  long long total=0;int totalCond=0;
  int i=0,j;down[0]=0;left[0]=0;right[0]=0;m[0]=0;bool computed=false;
  for(j=0;j<n/2;j++){
    bit=(1<<j);m[0]|=bit;
    down[1]=bit;left[1]=bit<<1;right[1]=bit>>1;
    m[1]=(down[1]|left[1]|right[1]);
    i=1;
    while(i>0){
      if((m[i]&mask)==mask){i--;}
      else{
        bit=(m[i]+1)&~m[i];m[i]|=bit;
        if((bit&mask)!=0){
          down[i+1]=down[i]|bit;left[i+1]=(left[i]|bit)<<1;right[i+1]=(right[i]|bit)>>1;
          m[i+1]=(down[i+1]|left[i+1]|right[i+1]);
          i++;
          if(i==mark){
            totalDown[totalCond]=down[i];totalLeft[totalCond]=left[i];totalRight[totalCond]=right[i];
            totalCond++;
            if(totalCond==steps){
              if(computed){
                hipMemcpy(results,resultsCuda,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int j=0;j<steps/THREAD_NUM;j++){total+=results[j];}
                computed=false;
              }
              hipMemcpy(downCuda,totalDown,sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              solve_nqueen_cuda_kernel_bt_bm<<<steps/THREAD_NUM,THREAD_NUM>>>(n,n-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              computed=true;totalCond=0;
            }
            i--;
          }
        }else{i --;}
      }
    }
  }
  if(computed){
    hipMemcpy(results,resultsCuda,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int j=0;j<steps/THREAD_NUM;j++){total+=results[j];}
    computed=false;
  }
  hipMemcpy(downCuda,totalDown,sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,sizeof(int)*totalCond,hipMemcpyHostToDevice);
  /** backTrack+bitmap*/
  solve_nqueen_cuda_kernel_bt_bm<<<steps/THREAD_NUM,THREAD_NUM>>>(n,n-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
  hipMemcpy(results,resultsCuda,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  for(int j=0;j<steps/THREAD_NUM;j++){total+=results[j];}	
  total*=2;
  if(n%2==1){
    computed=false;totalCond=0;bit=(1<<(n-1)/2);m[0]|=bit;
    down[1]=bit;left[1]=bit<<1;right[1]=bit>>1;
    m[1]=(down[1]|left[1]|right[1]);
    i=1;
    while(i>0){
      if((m[i]&mask)==mask){i--;}
      else{
        bit=(m[i]+1)&~m[i];m[i]|=bit;
        if((bit&mask)!=0){
          down[i+1]=down[i]|bit;left[i+1]=(left[i]|bit)<<1;right[i+1]=(right[i]|bit)>>1;
          m[i+1]=(down[i+1]|left[i+1]|right[i+1]);
          i++;
          if(i==mark){
            totalDown[totalCond]=down[i];totalLeft[totalCond]=left[i];totalRight[totalCond]=right[i];
            totalCond++;
            if(totalCond==steps){
              if(computed){
                hipMemcpy(results,resultsCuda,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int j=0;j<steps/THREAD_NUM;j++){total+=results[j];}
                computed=false;
              }
              hipMemcpy(downCuda,totalDown,sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              solve_nqueen_cuda_kernel_bt_bm<<<steps/THREAD_NUM,THREAD_NUM>>>(n,n-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              computed=true;totalCond=0;
            }
            i--;
          }
        }else{i --;}
      }
    }
    if(computed){
      hipMemcpy(results,resultsCuda,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
      for(int j=0;j<steps/THREAD_NUM;j++){total+=results[j];}
      computed=false;
    }
    hipMemcpy(downCuda,totalDown,sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(leftCuda,totalLeft,sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(rightCuda,totalRight,sizeof(int)*totalCond,hipMemcpyHostToDevice);
    /** backTrack+bitmap*/
    solve_nqueen_cuda_kernel_bt_bm<<<steps/THREAD_NUM,THREAD_NUM>>>(n,n-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
    hipMemcpy(results,resultsCuda,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int j=0;j<steps/THREAD_NUM;j++){total+=results[j];}
  }
  hipFree(downCuda);hipFree(leftCuda);hipFree(rightCuda);hipFree(resultsCuda);
  delete[] totalDown;delete[] totalLeft;delete[] totalRight;delete[] results;
  return total;
}
/** CUDA 初期化 **/
bool InitCUDA(){
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
void execCPU(int procNo){
  int min=4;int targetN=17;
  int msk;
  struct timeval t0;struct timeval t1;int ss;int ms;int dd;
  printf("\n%s\n"," N:          Total        Unique                 dd:hh:mm:ss.ms");
  for(int i=min;i<=targetN;i++){
    Total=Unique=C2=C4=C8=0;
    gettimeofday(&t0,NULL);   // 計測開始
    switch (procNo){
      case 1:
        for(int j=0;j<i;j++){ aBoard[j]=j; } //aBoardを初期化
        solve_nqueen_Recursive_BT(0,i);
        break;
      case 2:
        for(int j=0;j<i;j++){ aBoard[j]=-1; } //aBoardを初期化
        solve_nqueen_nonRecursive_BT(0,i);
        break;
      case 3:
        Total=solve_nqueen_Recursive_BT_BM(i,0,0,0);       
        break;
      case 4: 
        Total=solve_nqueen_nonRecursive_BT_BM(i);    
        break;
      case 5: 
        for(int j=0;j<i;j++){ aBoard[j]=j; } //aBoardを初期化
        msk=(1<<i)-1; // 初期化
        solve_nqueen_Recursive_BT_BM_SO(i,msk,0,0,0,0);
        Total=getTotal();
        Unique=getUnique();
        break;
      case 6: 
        for(int j=0;j<i;j++){ aBoard[j]=j; } //aBoardを初期化
        msk=(1<<i)-1; // 初期化
        Total=0;Unique=0;C2=0;C4=0;C8=0;
        solve_nqueen_nonRecursive_BT_BM_SO(i,msk,0,0,0,0);
        Total=getTotal();
        Unique=getUnique();
        break;
      case 7: 
        //Total=solve_nqueen_Recursive_BT_BM_SO_BOUND(i); 
        for(int j=0;j<i;j++){ aBoard[j]=j; } //aBoardを初期化
        msk=(1<<i)-1; // 初期化
        Total=0;Unique=0;C2=0;C4=0;C8=0;
        solve_nqueen_Recursive_BT_BM_SO_BOUND(i,msk); 
        Total=getTotal();
        Unique=getUnique();
        break;
      case 8: 
        //Unique=solve_nqueen_nonRecursive_BT_BM_SO_BOUND(i); 
        for(int j=0;j<i;j++){ aBoard[j]=j; } //aBoardを初期化
        msk=(1<<i)-1; // 初期化
        Total=0;Unique=0;C2=0;C4=0;C8=0;
        for(int BOUND1=0,BOUND2=i-1;BOUND1<i;BOUND1++,BOUND2--){
          solve_nqueen_nonRecursive_BT_BM_SO_BOUND(i,BOUND1,BOUND2,msk); 
        }
        Total=getTotal();
        Unique=getUnique();
        break;
      case 9:
        /* solve_nqueen_Recursive_BT_BM_SO_BOUND_BOUND2     */
        for(int j=0;j<i;j++){ aBoard[j]=j; } //aBoardを初期化
        msk=(1<<i)-1; // 初期化
        Total=0;Unique=0;C2=0;C4=0;C8=0;
        solve_nqueen_Recursive_BT_BM_SO_BOUND_BOUND2(i,msk); 
        Total=getTotal();
        Unique=getUnique();
        break ;
      case 10:
        /* solve_nqueen_nonRecursive_BT_BM_SO_BOUND_BOUND2     */
        for(int j=0;j<i;j++){ aBoard[j]=j; } //aBoardを初期化
        msk=(1<<i)-1; // 初期化
        Total=0;Unique=0;C2=0;C4=0;C8=0;
        solve_nqueen_nonRecursive_BT_BM_SO_BOUND_BOUND2(i,msk); 
        Total=getTotal();
        Unique=getUnique();
        break ;
      case 11:
        for(int j=0;j<i;j++){ aBoard[j]=j; } //aBoardを初期化
        msk=(1<<i)-1; // 初期化
        Total=0;Unique=0;C2=0;C4=0;C8=0;
        solve_nqueen_Recursive_BT_BM_SO_BOUND_BOUND2_OPT(i,msk);
        Total=getTotal();
        Unique=getUnique();
      break ;
      case 12:
        for(int j=0;j<i;j++){ aBoard[j]=j; } //aBoardを初期化
        msk=(1<<i)-1; // 初期化
        Total=0;Unique=0;C2=0;C4=0;C8=0;
        solve_nqueen_nonRecursive_BT_BM_SO_BOUND_BOUND2_OPT(i,msk);
        Total=getTotal();
        Unique=getUnique();
        break ;
      default: break;
    } 
    gettimeofday(&t1,NULL);   // 計測終了
    if (t1.tv_usec<t0.tv_usec) {
      dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
      ss=(t1.tv_sec-t0.tv_sec-1)%86400;
      ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
    } else {
      dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
      ss=(t1.tv_sec-t0.tv_sec)%86400;
      ms=(t1.tv_usec-t0.tv_usec+500)/10000;
    }
    int hh=ss/3600;
    int mm=(ss-hh*3600)/60;
    ss%=60;
    printf("%2d:%18ld%18ld%12.2d:%02d:%02d:%02d.%02d\n", i,Total,Unique,dd,hh,mm,ss,ms);
  }
}
int main(int argc,char** argv) {
  bool cpu=true,gpu=true;
  int argstart=1,steps=24576;
  /** パラメータの処理 */
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){gpu=false;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){cpu=false;}
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g] n steps\n",argv[0]);
    printf("  -c: CPU only\n");
    printf("  -g: GPU only\n");
    printf("Default to 8 queen\n");
  }
  /** 出力と実行 */
  /** CPU */
  if(cpu){
    printf("\n\n1. 再帰＋バックトラック(BT)");
    execCPU(1);  // solve_nqueen_Recursive_BT     
    printf("\n\n2. 非再帰＋バックトラック(BT)");
    execCPU(2);  // solve_nqueen_nonRecursive_BT     
    printf("\n\n3. 再帰＋バックトラック(BT)＋ビットマップ(BM)");
    execCPU(3);  // solve_nqueen_Recursive_BT_BM  
    printf("\n\n4. 非再帰＋バックトラック(BT)＋ビットマップ(BM)");
    execCPU(4);  // solve_nqueen_nonRecursive_BT_BM  
    printf("\n\n5. 再帰＋バックトラック(BT)＋ビットマップ(BM)＋対象解除法(SO)");
    execCPU(5);  // solve_nqueen_Recursive_BT_BM_SO     
    printf("\n\n6. 非再帰＋バックトラック(BT)＋ビットマップ(BM)＋対象解除法(SO)");
    execCPU(6);  // solve_nqueen_nonRecursive_BT_BM_SO     
    printf("\n\n7. 再帰＋バックトラック(BT)＋ビットマップ(BM)＋対象解除法(SO)＋枝刈り(BOUND)");
    execCPU(7);  // solve_nqueen_Recursive_BT_BM_SO_BOUND     
    printf("\n\n8. 非再帰＋バックトラック(BT)＋ビットマップ(BM)＋対象解除法(SO)＋枝刈り(BOUND)");
    execCPU(8);  // solve_nqueen_nonRecursive_BT_BM_SO_BOUND     
    printf("\n\n9. 再帰＋バックトラック(BT)＋ビットマップ(BM)＋対象解除法(SO)＋枝刈り(BOUND)＋BOUNDの枝刈り");
    execCPU(9);  // solve_nqueen_Recursive_BT_BM_SO_BOUND_BOUND2     
    printf("\n\n10. 非再帰＋バックトラック(BT)＋ビットマップ(BM)＋対象解除法(SO)＋枝刈り(BOUND)＋BOUNDの枝刈り");
    execCPU(10);  // solve_nqueen_nonRecursive_BT_BM_SO_BOUND_BOUND2     
    printf("\n\n11. 再帰＋バックトラック(BT)＋ビットマップ(BM)＋対象解除法(SO)＋枝刈り(BOUND)＋BOUNDの枝刈り＋最適化");
    execCPU(11);  // solve_nqueen_nonRecursive_BT_BM_SO_BOUND_BOUND2_OPT     
    printf("\n\n12. 非再帰＋バックトラック(BT)＋ビットマップ(BM)＋対象解除法(SO)＋枝刈り(BOUND)＋BOUNDの枝刈り＋最適化");
    execCPU(12);  // solve_nqueen_nonRecursive_BT_BM_SO_BOUND_BOUND2_OPT     
  }
  /** GPU */
  if(gpu){
    if(!InitCUDA()){return 0;}
    int min=4;int targetN=17;
    struct timeval t0;struct timeval t1;int ss;int ms;int dd;
    printf("%s\n"," N:          Total        Unique                 dd:hh:mm:ss.ms");
    for(int i=min;i<=targetN;i++){
      gettimeofday(&t0,NULL);   // 計測開始
      Total=solve_nqueen_cuda(i,steps);
      gettimeofday(&t1,NULL);   // 計測終了
      if (t1.tv_usec<t0.tv_usec) {
        dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      } else {
        dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%18ld%18ld%12.2d:%02d:%02d:%02d.%02d\n", i,Total,Unique,dd,hh,mm,ss,ms);
    }
  }
  return 0;
}


