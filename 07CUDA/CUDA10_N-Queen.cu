/**
 CUDAで学ぶアルゴリズムとデータ構造
 ステップバイステップでＮ−クイーン問題を最適化
 一般社団法人  共同通信社  情報技術局  鈴木  維一郎(suzuki.iichiro@kyodonews.jp)

 コンパイルと実行
 $ nvcc CUDA**_N-Queen.cu && ./a.out (-c|-r|-g|-s)
                    -c:cpu 
                    -r cpu再帰 
                    -g GPU 
                    -s SGPU(サマーズ版と思われる)

 １０．クイーンの位置による分岐BOUND1,2

  前章のコードは全ての解を求めた後に、ユニーク解以外の対称解を除去していた
  ある意味、「生成検査法（generate ＆ test）」と同じである
  問題の性質を分析し、バックトラッキング/前方検査法と同じように、無駄な探索を省略することを考える
  ユニーク解に対する左右対称解を予め削除するには、1行目のループのところで、
  右半分だけにクイーンを配置するようにすればよい
  Nが奇数の場合、クイーンを1行目中央に配置する解は無い。
  他の3辺のクィーンが中央に無い場合、その辺が上辺に来るよう回転し、場合により左右反転することで、
  最小値解とすることが可能だから、中央に配置したものしかユニーク解には成り得ない
  しかし、上辺とその他の辺の中央にクィーンは互いの効きになるので、配置することが出来ない


  1. １行目角にクイーンがある場合、とそうでない場合で処理を分ける
    １行目かどうかの条件判断はループ外に出してもよい
    処理時間的に有意な差はないので、分かりやすいコードを示した
  2.１行目角にクイーンがある場合、回転対称形チェックを省略することが出来る
    １行目角にクイーンがある場合、他の角にクイーンを配置することは不可
    鏡像についても、主対角線鏡像のみを判定すればよい
    ２行目、２列目を数値とみなし、２行目＜２列目という条件を課せばよい

  １行目角にクイーンが無い場合、クイーン位置より右位置の８対称位置にクイーンを置くことはできない
  置いた場合、回転・鏡像変換により得られる状態のユニーク判定値が明らかに大きくなる
    ☓☓・・・Ｑ☓☓
    ☓・・・／｜＼☓
    ｃ・・／・｜・rt
    ・・／・・｜・・
    ・／・・・｜・・
    lt・・・・｜・ａ
    ☓・・・・｜・☓
    ☓☓ｂ・・dn☓☓
    
  １行目位置が確定した時点で、配置可能位置を計算しておく（☓の位置）
  lt, dn, lt 位置は効きチェックで配置不可能となる
  回転対称チェックが必要となるのは、クイーンがａ, ｂ, ｃにある場合だけなので、
  90度、180度、270度回転した状態のユニーク判定値との比較を行うだけで済む
 *

 実行結果

$ nvcc CUDA10_N-Queen.cu  && ./a.out -r
１０．CPUR 再帰 クイーンの位置による分岐BOUND1,2
 N:        Total       Unique        hh:mm:ss.ms
 4:            2               1            0.00
 5:           10               2            0.00
 6:            4               1            0.00
 7:           40               6            0.00
 8:           92              12            0.00
 9:          352              46            0.00
10:          724              92            0.00
11:         2680             341            0.00
12:        14200            1787            0.01
13:        73712            9233            0.05
14:       365596           45752            0.31
15:      2279184          285053            1.94
16:     14772512         1846955           14.54
17:     95815104        11977939         1:38.93

$ nvcc CUDA10_N-Queen.cu  && ./a.out -c
１０．CPU 非再帰 クイーンの位置による分岐BOUND1,2
 N:        Total       Unique        hh:mm:ss.ms
 4:            2               1            0.00
 5:           10               2            0.00
 6:            4               1            0.00
 7:           40               6            0.00
 8:           92              12            0.00
 9:          352              46            0.00
10:          724              92            0.00
11:         2680             341            0.00
12:        14200            1787            0.01
13:        73712            9233            0.05
14:       365596           45752            0.28
15:      2279184          285053            1.80
16:     14772512         1846955           13.55
17:     95815104        11977939         1:33.01

$ nvcc CUDA10_N-Queen.cu  && ./a.out -g
１０．GPU 非再帰 クイーンの位置による分岐BOUND1,2
 N:        Total      Unique      dd:hh:mm:ss.ms
 4:            2               1  00:00:00:00.03
 5:           10               2  00:00:00:00.00
 6:            4               1  00:00:00:00.00
 7:           40               6  00:00:00:00.00
 8:           92              12  00:00:00:00.00
 9:          352              46  00:00:00:00.02
10:          724              92  00:00:00:00.05
11:         2680             341  00:00:00:00.16
12:        14200            1787  00:00:00:00.69
13:        73712            9233  00:00:00:03.22
14:       365596           45752  00:00:00:20.86
15:       278064           34758  00:00:00:22.82

*/
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define THREAD_NUM		96
#define MAX 27
//変数宣言
long Total=0 ;      //GPU
long Unique=0;      //GPU
int aBoard[MAX];
int aT[MAX];
int aS[MAX];
int COUNT2,COUNT4,COUNT8;
int BOUND1,BOUND2,TOPBIT,ENDBIT,SIDEMASK,LASTMASK;
//
//関数宣言 GPU
__device__
int symmetryOps_bitmap(int si,int *d_aBoard,int *d_aT,int *d_aS);
__global__
void nqueen_cuda_backTrack2(long *d_results,int *d_aBoard,int *d_aT,int *d_aS,long UNIQUE,int size,int mask,int row,int left,int down,int right);
__global__
void nqueen_cuda_backTrack1(long *d_results,int *d_aBoard,int *d_aT,int *d_aS,long UNIQUE,int size,int mask,int row,int left,int down,int right);
void solve_nqueen_cuda(int si,int mask,long results[2],int steps);
//関数宣言 SGPU
__global__ void sgpu_cuda_kernel(int size,int mark,unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* results,int totalCond);
long long sgpu_solve_nqueen_cuda(int size,int steps);
//関数宣言 CPU/GPU
__device__ __host__ void rotate_bitmap(int bf[],int af[],int si);
__device__ __host__ void vMirror_bitmap(int bf[],int af[],int si);
__device__ __host__ int intncmp(int lt[],int rt[],int n);
__device__ __host__ int rh(int a,int sz);
//関数宣言 CPU/CPUR
void TimeFormat(clock_t utime,char *form);
long getUnique();
long getTotal();
void symmetryOps_bitmap(int si);
//関数宣言 CPU 非再帰版
void backTrack2(int si,int mask,int y,int l,int d,int r);
void backTrack1(int si,int mask,int y,int l,int d,int r);
void NQueen(int size,int mask);
//関数宣言 CPUR 再帰版
void backTrackR2(int si,int mask,int y,int l,int d,int r);
void backTrackR1(int si,int mask,int y,int l,int d,int r);
void NQueenR(int size,int mask);
//関数宣言【通常版】
void backTrack2D_NR(int si,int mask,int y,int l,int d,int r);
void backTrack1D_NR(int si,int mask,int y,int l,int d,int r);
void NQueenD(int size,int mask);
void backTrack2D(int si,int mask,int y,int l,int d,int r);
void backTrack1D(int si,int mask,int y,int l,int d,int r);
void NQueenDR(int size,int mask);
//
// GPU
__device__
int symmetryOps_bitmap(int si,int *d_aBoard,int *d_aT,int *d_aS){
  int nEquiv;
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<si;i++){ d_aT[i]=d_aBoard[i];}
  rotate_bitmap(d_aT,d_aS,si);    //時計回りに90度回転
  int k=intncmp(d_aBoard,d_aS,si);
  if(k>0)return 0;
  if(k==0){ nEquiv=2;}else{
    rotate_bitmap(d_aS,d_aT,si);  //時計回りに180度回転
    k=intncmp(d_aBoard,d_aT,si);
    if(k>0)return 0;
    if(k==0){ nEquiv=4;}else{
      rotate_bitmap(d_aT,d_aS,si);//時計回りに270度回転
      k=intncmp(d_aBoard,d_aS,si);
      if(k>0){ return 0;}
      nEquiv=8;
    }
  }
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<si;i++){ d_aS[i]=d_aBoard[i];}
  vMirror_bitmap(d_aS,d_aT,si);   //垂直反転
  k=intncmp(d_aBoard,d_aT,si);
  if(k>0){ return 0; }
  if(nEquiv>2){             //-90度回転 対角鏡と同等
    rotate_bitmap(d_aT,d_aS,si);
    k=intncmp(d_aBoard,d_aS,si);
    if(k>0){return 0;}
    if(nEquiv>4){           //-180度回転 水平鏡像と同等
      rotate_bitmap(d_aS,d_aT,si);
      k=intncmp(d_aBoard,d_aT,si);
      if(k>0){ return 0;}       //-270度回転 反対角鏡と同等
      rotate_bitmap(d_aT,d_aS,si);
      k=intncmp(d_aBoard,d_aS,si);
      if(k>0){ return 0;}
    }
  }
  return nEquiv;
}
// GPU
__global__
void nqueen_cuda_backTrack2(long *d_results,int *d_aBoard,int *d_aT,int *d_aS,long UNIQUE,int size,int mask,int row,int left,int down,int right){
  int bitmap,bit;
  int b[100], *p=b;
  int odd=size&1; //奇数:1 偶数:0
  for(int i=0;i<(1+odd);++i){
    bitmap=0;
    if(0==i){
      int half=size>>1; // size/2
      bitmap=(1<<half)-1;
    }else{
      bitmap=1<<(size>>1);
    }
    mais1:bitmap=mask&~(left|down|right);
    if(row==size){
      if(!bitmap){
        d_aBoard[row]=bitmap;
        int s=symmetryOps_bitmap(size,d_aBoard,d_aT,d_aS);
        if(s!=0){
        //print(size); //print()でTOTALを++しない
        //ホストに戻す配列にTOTALを入れる
        //スレッドが１つの場合は配列は１個
          d_results[1]++;
          d_results[0]+=s;   //対称解除で得られた解数を加算
        }
      }
    }else{
      if(bitmap){
        outro:bitmap^=d_aBoard[row]=bit=-bitmap&bitmap;
        if(bitmap){
         *p++=left;
         *p++=down;
         *p++=right;
        }
        *p++=bitmap;
        row++;
        left=(left|bit)<<1;
        down=down|bit;
        right=(right|bit)>>1;
        goto mais1;
        //Backtrack2(y+1, (left | bit)<<1, down | bit, (right | bit)>>1);
        volta:if(p<=b)
        return;
        row--;
        bitmap=*--p;
        if(bitmap){
          right=*--p;
          down=*--p;
          left=*--p;
          goto outro;
        }else{
          goto volta;
        }
      }
    }
    goto volta;
  }
}
// GPU
__global__
void nqueen_cuda_backTrack1(long *d_results,int *d_aBoard,int *d_aT,int *d_aS,long UNIQUE,int size,int mask,int row,int left,int down,int right){
  int bitmap,bit;
  int b[100], *p=b;
  int sizeE=size-1;
  int odd=size&1; //奇数:1 偶数:0
  for(int i=0;i<(1+odd);++i){
    bitmap=0;
    if(0==i){
      int half=size>>1; // size/2
      bitmap=(1<<half)-1;
    }else{
      bitmap=1<<(size>>1);
    }
    b1mais1:bitmap=mask&~(left|down|right);
    if(row==sizeE){
      if(bitmap){
        d_aBoard[row]=bitmap;
        int s=symmetryOps_bitmap(size,d_aBoard,d_aT,d_aS); 
        if(s!=0){
        //print(size); //print()でTOTALを++しない
        //ホストに戻す配列にTOTALを入れる
        //スレッドが１つの場合は配列は１個
          d_results[1]++;
          d_results[0]+=s;   //対称解除で得られた解数を加算
        }
      }
    }else{
      if(bitmap){
        b1outro:bitmap^=d_aBoard[row]=bit=-bitmap&bitmap;
        if(bitmap){
          *p++=left;
          *p++=down;
          *p++=right;
        }
        *p++=bitmap;
        row++;
        left=(left|bit)<<1;
        down=down|bit;
        right=(right|bit)>>1;
        goto b1mais1;
        //Backtrack1(y+1, (left | bit)<<1, down | bit, (right | bit)>>1);
        b1volta:if(p<=b)
        return;
        row--;
        bitmap=*--p;
        if(bitmap){
          right=*--p;
          down=*--p;
          left=*--p;
          goto b1outro;
        }else{
          goto b1volta;
        }
      }
    }
    goto b1volta;
  }
}
// GPU
void solve_nqueen_cuda(int si,int mask,long results[2],int steps){
    int BOUND1,BOUND2,TOPBIT,ENDBIT,SIDEMASK,LASTMASK;
    //メモリ登録
    long *h_results;
    int *h_aBoard;
    int *h_aT;
    int *h_aS;
    hipHostMalloc((void**)&h_results,sizeof(long)*steps, hipHostMallocDefault);
    hipHostMalloc((void**)&h_aBoard,sizeof(int)*MAX, hipHostMallocDefault);
    hipHostMalloc((void**)&h_aT,sizeof(int)*MAX, hipHostMallocDefault);
    hipHostMalloc((void**)&h_aS,sizeof(int)*MAX, hipHostMallocDefault);
    long *d_results;
    int *d_aBoard;
    int *d_aT;
    int *d_aS;
    hipMalloc((void**)&d_results,sizeof(long)*steps);
    hipMalloc((void**)&d_aBoard,sizeof(int)*MAX);
    hipMalloc((void**)&d_aT,sizeof(int)*MAX);
    hipMalloc((void**)&d_aS,sizeof(int)*MAX);
    //ロジック
    int bit;
    TOPBIT=1<<(si-1);
    h_aBoard[0]=1;
    for(BOUND1=2;BOUND1<si-1;BOUND1++){
      h_aBoard[1]=bit=(1<<BOUND1);
      //host to device
      hipMemcpy(d_aBoard,h_aBoard,
          sizeof(int)*MAX,hipMemcpyHostToDevice);
      hipMemcpy(d_aT,h_aT,
          sizeof(int)*MAX,hipMemcpyHostToDevice);
      hipMemcpy(d_aS,h_aS,
          sizeof(int)*MAX,hipMemcpyHostToDevice);
      hipMemcpy(d_results,h_results,
          sizeof(int)*steps,hipMemcpyHostToDevice);
      //実行  
      nqueen_cuda_backTrack1<<<1,1>>>(d_results,d_aBoard,d_aT,d_aS,0,si,mask,2,(2|bit)<<1,(1|bit),(bit>>1));
      //host to device
      hipMemcpy(h_results,d_results,
          sizeof(long)*steps,hipMemcpyDeviceToHost);
    }
    SIDEMASK=LASTMASK=(TOPBIT|1);
    ENDBIT=(TOPBIT>>1);
    for(BOUND1=1,BOUND2=si-2;BOUND1<BOUND2;BOUND1++,BOUND2--){
      h_aBoard[0]=bit=(1<<BOUND1);
      //host to device
      hipMemcpy(d_aBoard,h_aBoard,
          sizeof(int)*MAX,hipMemcpyHostToDevice);
      hipMemcpy(d_aT,h_aT,
          sizeof(int)*MAX,hipMemcpyHostToDevice);
      hipMemcpy(d_aS,h_aS,
          sizeof(int)*MAX,hipMemcpyHostToDevice);
      hipMemcpy(d_results,h_results,
          sizeof(long)*steps,hipMemcpyHostToDevice);
      //実行
      nqueen_cuda_backTrack2<<<1,1>>>(d_results,d_aBoard,d_aT,d_aS,h_results[1],si,mask,1,bit<<1,bit,bit>>1);
      //device to host
      hipMemcpy(h_results,d_results,
          sizeof(long)*steps,hipMemcpyDeviceToHost);
      LASTMASK|=LASTMASK>>1|LASTMASK<<1;
      ENDBIT>>=1;
    }
    //解を代入
    results[0]=h_results[0];
    results[1]=h_results[1];
    //メモリ解放
    hipHostFree(h_results);
    hipHostFree(h_aBoard);
    hipHostFree(h_aT);
    hipHostFree(h_aS);
    hipFree(d_aBoard);
    hipFree(d_aT);
    hipFree(d_aS);
    hipFree(d_results);
}
// SGPU
__global__ void sgpu_cuda_kernel(int size,int mark,unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,unsigned int* results,int totalCond){
  const int tid=threadIdx.x;
  const int bid=blockIdx.x;
  const int idx=bid*blockDim.x+tid;
  __shared__ unsigned int down[THREAD_NUM][10];
  __shared__ unsigned int left[THREAD_NUM][10];
  __shared__ unsigned int right[THREAD_NUM][10];
  __shared__ unsigned int bitmap[THREAD_NUM][10];
  __shared__ unsigned int sum[THREAD_NUM];
  const unsigned int mask=(1<<size)-1;
  int total=0;
  int row=0;
  unsigned int bit;
  if(idx<totalCond){
    down[tid][row]=totalDown[idx];
    left[tid][row]=totalLeft[idx];
    right[tid][row]=totalRight[idx];
    bitmap[tid][row]=down[tid][row]|left[tid][row]|right[tid][row];
    while(row>=0){
      if((bitmap[tid][row]&mask)==mask){row--;}
      else{
        bit=(bitmap[tid][row]+1)&~bitmap[tid][row];
        bitmap[tid][row]|=bit;
        if((bit&mask)!=0){
          if(row+1==mark){total++;row--;}
          else{
            down[tid][row+1]=down[tid][row]|bit;
            left[tid][row+1]=(left[tid][row]|bit)<<1;
            right[tid][row+1]=(right[tid][row]|bit)>>1;
            bitmap[tid][row+1]=(down[tid][row+1]|left[tid][row+1]|right[tid][row+1]);
            row++;
          }
        }else{row--;}
      }
    }
    sum[tid]=total;
  }else{sum[tid]=0;} 
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){sum[tid]+=sum[tid+64];} 
  __syncthreads();if(tid<32){sum[tid]+=sum[tid+32];} 
  __syncthreads();if(tid<16){sum[tid]+=sum[tid+16];} 
  __syncthreads();if(tid<8){sum[tid]+=sum[tid+8];} 
  __syncthreads();if(tid<4){sum[tid]+=sum[tid+4];} 
  __syncthreads();if(tid<2){sum[tid]+=sum[tid+2];} 
  __syncthreads();if(tid<1){sum[tid]+=sum[tid+1];} 
  __syncthreads();if(tid==0){results[bid]=sum[0];}
}
// SGPU
long long sgpu_solve_nqueen_cuda(int size,int steps) {
  unsigned int down[32];
  unsigned int left[32];
  unsigned int right[32];
  unsigned int bitmap[32];
  unsigned int bit;
  if(size<=0||size>32){return 0;}
  unsigned int* totalDown=new unsigned int[steps];
  unsigned int* totalLeft=new unsigned int[steps];
  unsigned int* totalRight=new unsigned int[steps];
  unsigned int* results=new unsigned int[steps];
  unsigned int* downCuda;
  unsigned int* leftCuda;
  unsigned int* rightCuda;
  unsigned int* resultsCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  const unsigned int mask=(1<<size)-1;
  const unsigned int mark=size>11?size-10:2;
  long long total=0;
  int totalCond=0;
  int row=0;
  down[0]=0;
  left[0]=0;
  right[0]=0;
  bitmap[0]=0;
  bool matched=false;
  for(int col=0;col<size/2;col++){
    bit=(1<<col);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM>>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
  }
  if(matched){
    hipMemcpy(results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
    matched=false;
  }
  hipMemcpy(downCuda,totalDown,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,
      sizeof(int)*totalCond,hipMemcpyHostToDevice);
  /** backTrack+bitmap*/
  sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM>>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
  hipMemcpy(results,resultsCuda,
      sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}	
  total*=2;


  if(size%2==1){
    matched=false;
    totalCond=0;
    bit=(1<<(size-1)/2);
    bitmap[0]|=bit;
    down[1]=bit;
    left[1]=bit<<1;
    right[1]=bit>>1;
    bitmap[1]=(down[1]|left[1]|right[1]);
    row=1;
    while(row>0){
      if((bitmap[row]&mask)==mask){row--;}
      else{
        bit=(bitmap[row]+1)&~bitmap[row];
        bitmap[row]|=bit;
        if((bit&mask)!=0){
          down[row+1]=down[row]|bit;
          left[row+1]=(left[row]|bit)<<1;
          right[row+1]=(right[row]|bit)>>1;
          bitmap[row+1]=(down[row+1]|left[row+1]|right[row+1]);
          row++;
          if(row==mark){
            totalDown[totalCond]=down[row];
            totalLeft[totalCond]=left[row];
            totalRight[totalCond]=right[row];
            totalCond++;
            if(totalCond==steps){
              if(matched){
                hipMemcpy(results,resultsCuda,
                    sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
                matched=false;
              }
              hipMemcpy(downCuda,totalDown,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,
                  sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM>>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              matched=true;
              totalCond=0;
            }
            row--;
          }
        }else{row--;}
      }
    }
    if(matched){
      hipMemcpy(results,resultsCuda,
          sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
      for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
      matched=false;
    }
    hipMemcpy(downCuda,totalDown,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(leftCuda,totalLeft,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(rightCuda,totalRight,
        sizeof(int)*totalCond,hipMemcpyHostToDevice);
    /** backTrack+bitmap*/
    sgpu_cuda_kernel<<<steps/THREAD_NUM,THREAD_NUM>>>(size,size-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
    hipMemcpy(results,resultsCuda,
        sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int col=0;col<steps/THREAD_NUM;col++){total+=results[col];}
  }
  hipFree(downCuda);
  hipFree(leftCuda);
  hipFree(rightCuda);
  hipFree(resultsCuda);
  delete[] totalDown;
  delete[] totalLeft;
  delete[] totalRight;
  delete[] results;
  return total;
}
/** CUDA 初期化 **/
bool InitCUDA(){
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
//hh:mm:ss.ms形式に処理時間を出力
void TimeFormat(clock_t utime,char *form){
  int dd,hh,mm;
  float ftime,ss;
  ftime=(float)utime/CLOCKS_PER_SEC;
  mm=(int)ftime/60;
  ss=ftime-(int)(mm*60);
  dd=mm/(24*60);
  mm=mm%(24*60);
  hh=mm/60;
  mm=mm%60;
  if(dd)
    sprintf(form,"%4d %02d:%02d:%05.2f",dd,hh,mm,ss);
  else if(hh)
    sprintf(form,"     %2d:%02d:%05.2f",hh,mm,ss);
  else if(mm)
    sprintf(form,"        %2d:%05.2f",mm,ss);
  else
    sprintf(form,"           %5.2f",ss);
}
//
__device__ __host__
int rh(int a,int size){
  int tmp=0;
  for(int i=0;i<=size;i++){
    if(a&(1<<i)){
      return tmp|=(1<<(size-i));
    }
  }
  return tmp;
}
//
__device__ __host__
void vMirror_bitmap(int bf[],int af[],int size){
  int score;
  for(int i=0;i<size;i++){
    score=bf[i];
    af[i]=rh(score,size-1);
  }
}
//
__device__ __host__
void rotate_bitmap(int bf[],int af[],int size){
  int t;
  for(int i=0;i<size;i++){
    t=0;
    for(int j=0;j<size;j++){
      t|=((bf[j]>>i)&1)<<(size-j-1); // x[j] の i ビット目を
    }
    af[i]=t;                        // y[i] の j ビット目にする
  }
}
//
__device__ __host__
int intncmp(int lt[],int rt[],int n){
  int rtn=0;
  for(int k=0;k<n;k++){
    rtn=lt[k]-rt[k];
    if(rtn!=0){
      break;
    }
  }
  return rtn;
}
//
long getUnique(){
  return COUNT2+COUNT4+COUNT8;
}
//
long getTotal(){
  return COUNT2*2+COUNT4*4+COUNT8*8;
}
//
void symmetryOps_bitmap(int size){
  int nEquiv;
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<size;i++){
    aT[i]=aBoard[i];
  }
  rotate_bitmap(aT,aS,size);    //時計回りに90度回転
  int k=intncmp(aBoard,aS,size);
  if(k>0) return;
  if(k==0){
    nEquiv=2;
  }else{
    rotate_bitmap(aS,aT,size);  //時計回りに180度回転
    k=intncmp(aBoard,aT,size);
    if(k>0) return;
    if(k==0){
      nEquiv=4;
    }else{
      rotate_bitmap(aT,aS,size);  //時計回りに270度回転
      k=intncmp(aBoard,aS,size);
      if(k>0){
        return;
      }
      nEquiv=8;
    }
  }
  // 回転・反転・対称チェックのためにboard配列をコピー
  for(int i=0;i<size;i++){
    aS[i]=aBoard[i];
  }
  vMirror_bitmap(aS,aT,size);   //垂直反転
  k=intncmp(aBoard,aT,size);
  if(k>0){
    return;
  }
  if(nEquiv>2){             //-90度回転 対角鏡と同等
    rotate_bitmap(aT,aS,size);
    k=intncmp(aBoard,aS,size);
    if(k>0){
      return;
    }
    if(nEquiv>4){           //-180度回転 水平鏡像と同等
      rotate_bitmap(aS,aT,size);
      k=intncmp(aBoard,aT,size);
      if(k>0){
        return;
      }       //-270度回転 反対角鏡と同等
      rotate_bitmap(aT,aS,size);
      k=intncmp(aBoard,aS,size);
      if(k>0){
        return;
      }
    }
  }
  if(nEquiv==2){ COUNT2++; }
  if(nEquiv==4){ COUNT4++; }
  if(nEquiv==8){ COUNT8++; }
}
//
//CPU 非再帰版 ロジックメソッド
void backTrack2(int size,int mask, int row,int h_left,int h_down,int h_right){
	unsigned int left[size];
    unsigned int down[size];
	unsigned int right[size];
    unsigned int bitmap[size];
	left[row]=h_left;
	down[row]=h_down;
	right[row]=h_right;
	bitmap[row]=mask&~(left[row]|down[row]|right[row]);
    unsigned int bit;
    unsigned int sizeE=size-1;
    int mark=row;
    //固定していれた行より上はいかない
    while(row>=mark){//row=1 row>=1, row=2 row>=2
      if(bitmap[row]==0){
        --row;
      }else{
        bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]); 
        if((bit&mask)!=0){
          if(row==sizeE){
            symmetryOps_bitmap(size);
            --row;
          }else{
            int n=row++;
            left[row]=(left[n]|bit)<<1;
            down[row]=down[n]|bit;
            right[row]=(right[n]|bit)>>1;
            bitmap[row]=mask&~(left[row]|down[row]|right[row]);
          }
        }else{
           --row;
        }
      }  
    }
}
//
void backTrack1(int size,int mask, int row,int h_left,int h_down,int h_right){
	unsigned int left[size];
    unsigned int down[size];
	unsigned int right[size];
    unsigned int bitmap[size];
	left[row]=h_left;
	down[row]=h_down;
	right[row]=h_right;
	bitmap[row]=mask&~(left[row]|down[row]|right[row]);
    unsigned int bit;
    unsigned int sizeE=size-1;
    int mark=row;
    //固定していれた行より上はいかない
    while(row>=mark){//row=1 row>=1, row=2 row>=2
      if(bitmap[row]==0){
        --row;
      }else{
        bitmap[row]^=aBoard[row]=bit=(-bitmap[row]&bitmap[row]); 
        if((bit&mask)!=0){
          if(row==sizeE){
            symmetryOps_bitmap(size);
            --row;
          }else{
            int n=row++;
            left[row]=(left[n]|bit)<<1;
            down[row]=down[n]|bit;
            right[row]=(right[n]|bit)>>1;
            bitmap[row]=mask&~(left[row]|down[row]|right[row]);
          }
        }else{
           --row;
        }
      }  
    }
}
void NQueen(int size,int mask){
  int bit=0;
  //10では枝借りはまだしないのでTOPBIT,SIDEMASK,LASTMASK,ENDBITは使用しない
  //backtrack1
  //1行め右端 0
  int col=0;
  aBoard[0]=bit=(1<<col);
  int left=bit<<1;
  int down=bit;
  int right=bit>>1;
  //2行目は右から3列目から左端から2列目まで
  for(int col_j=2;col_j<size-1;col_j++){
      aBoard[1]=bit=(1<<col_j);
      backTrack1(size,mask,2,(left|bit)<<1,(down|bit),(right|bit)>>1);
  }
  //backtrack2
  //1行目右から2列目から
  //偶数個は1/2 n=8 なら 1,2,3 奇数個は1/2+1 n=9 なら 1,2,3,4
  for(int col=1,col2=size-2;col<col2;col++,col2--){
      aBoard[0]=bit=(1<<col);
      backTrack2(size,mask,1,bit<<1,bit,bit>>1);
  }
}
//
void backTrackR1(int size,int mask, int row,int left,int down,int right){
 int bitmap=0;
 int bit=0;
 int sizeE=size-1;
 bitmap=(mask&~(left|down|right));
 if(row==sizeE){
   if(bitmap){
     aBoard[row]=(-bitmap&bitmap);
     symmetryOps_bitmap(size);
   }
  }else{
    while(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap);
      backTrackR1(size,mask,row+1,(left|bit)<<1, down|bit,(right|bit)>>1);
    }
  }
}
//
void backTrackR2(int size,int mask, int row,int left,int down,int right){
 int bitmap=0;
 int bit=0;
 int sizeE=size-1;
 bitmap=(mask&~(left|down|right));
 if(row==sizeE){
   if(bitmap){
     aBoard[row]=(-bitmap&bitmap);
     symmetryOps_bitmap(size);
   }
  }else{
    while(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap);
      backTrackR2(size,mask,row+1,(left|bit)<<1, down|bit,(right|bit)>>1);
    }
  }
}
//CPUR 再帰版 ロジックメソッド
void NQueenR(int size,int mask){
  int bit=0;
  //10では枝借りはまだしないのでTOPBIT,SIDEMASK,LASTMASK,ENDBITは使用しない
  //backtrack1
  //1行め右端 0
  int col=0;
  aBoard[0]=bit=(1<<col);
  int left=bit<<1;
  int down=bit;
  int right=bit>>1;
  //2行目は右から3列目から左端から2列目まで
  for(int col_j=2;col_j<size-1;col_j++){
      aBoard[1]=bit=(1<<col_j);
      backTrackR1(size,mask,2,(left|bit)<<1,(down|bit),(right|bit)>>1);
  }
  //backtrack2
  //1行目右から2列目から
  //偶数個は1/2 n=8 なら 1,2,3 奇数個は1/2+1 n=9 なら 1,2,3,4
  for(int col=1,col2=size-2;col<col2;col++,col2--){
      aBoard[0]=bit=(1<<col);
      backTrackR2(size,mask,1,bit<<1,bit,bit>>1);
  }
}
//CPU 非再帰版 backTrack2
void backTrack2D_NR(int size,int mask,int row,int left,int down,int right){
	int bitmap,bit;
	int b[100], *p=b;
  int odd=size&1; //奇数:1 偶数:0
  for(int i=0;i<(1+odd);++i){
    bitmap=0;
    if(0==i){
      int half=size>>1; // size/2
      bitmap=(1<<half)-1;
    }else{
      bitmap=1<<(size>>1);
      // down[1]=bitmap;
      // right[1]=(bitmap>>1);
      // left[1]=(bitmap<<1);
      // pnStack=aStack+1;
      // *pnStack++=0;
    }
    mais1:bitmap=mask&~(left|down|right);
    if(row==size){
      if(!bitmap){
        aBoard[row]=bitmap;
        symmetryOps_bitmap(size);
      }
    }else{
      if(bitmap){
        outro:bitmap^=aBoard[row]=bit=-bitmap&bitmap;
        if(bitmap){
          *p++=left;
          *p++=down;
          *p++=right;
        }
        *p++=bitmap;
        row++;
        left=(left|bit)<<1;
        down=down|bit;
        right=(right|bit)>>1;
        goto mais1;
        //Backtrack2(y+1, (left | bit)<<1, down | bit, (right | bit)>>1);
        volta:if(p<=b)
          return;
        row--;
        bitmap=*--p;
        if(bitmap){
          right=*--p;
          down=*--p;
          left=*--p;
          goto outro;
        }else{
          goto volta;
        }
      }
    }
    goto volta;
  }
}
//CPU 非再帰版 backTrack1
void backTrack1D_NR(int size,int mask,int row,int left,int down,int right){
  int bitmap,bit;
  int b[100], *p=b;
  int sizeE=size-1;
  int odd=size&1; //奇数:1 偶数:0
  for(int i=0;i<(1+odd);++i){
    bitmap=0;
    if(0==i){
      int half=size>>1; // size/2
      bitmap=(1<<half)-1;
    }else{
      bitmap=1<<(size>>1);
      // down[1]=bitmap;
      // right[1]=(bitmap>>1);
      // left[1]=(bitmap<<1);
      // pnStack=aStack+1;
      // *pnStack++=0;
    }
    b1mais1:bitmap=mask&~(left|down|right);
    if(row==sizeE){
      if(bitmap){
        aBoard[row]=bitmap;
        symmetryOps_bitmap(size);
      }
    }else{
      if(bitmap){
        b1outro:bitmap^=aBoard[row]=bit=-bitmap&bitmap;
        if(bitmap){
          *p++=left;
          *p++=down;
          *p++=right;
        }
        *p++=bitmap;
        row++;
        left=(left|bit)<<1;
        down=down|bit;
        right=(right|bit)>>1;
        goto b1mais1;
        //Backtrack1(y+1, (left | bit)<<1, down | bit, (right | bit)>>1);
        b1volta:if(p<=b)
          return;
        row--;
        bitmap=*--p;
        if(bitmap){
          right=*--p;
          down=*--p;
          left=*--p;
          goto b1outro;
        }else{
          goto b1volta;
        }
      }
    }
    goto b1volta;
  }
}
//CPU 非再帰版 ロジックメソッド
void NQueenD(int size,int mask){
  int bit;
  TOPBIT=1<<(size-1);
  aBoard[0]=1;
  for(BOUND1=2;BOUND1<size-1;BOUND1++){
    aBoard[1]=bit=(1<<BOUND1);
    //backTrack1(size,mask,2,(2|bit)<<1,(1|bit),(bit>>1));
    backTrack1D_NR(size,mask,2,(2|bit)<<1,(1|bit),(bit>>1));
  }
  SIDEMASK=LASTMASK=(TOPBIT|1);
  ENDBIT=(TOPBIT>>1);
  for(BOUND1=1,BOUND2=size-2;BOUND1<BOUND2;BOUND1++,BOUND2--){
    aBoard[0]=bit=(1<<BOUND1);
    //backTrack1(size,mask,1,bit<<1,bit,bit>>1);
    backTrack2D_NR(size,mask,1,bit<<1,bit,bit>>1);
    LASTMASK|=LASTMASK>>1|LASTMASK<<1;
    ENDBIT>>=1;
  }
}
//
void backTrack2D(int size,int mask,int row,int left,int down,int right){
  int bit;
  int bitmap=mask&~(left|down|right); /* 配置可能フィールド */
  if(row==size){
    aBoard[row]=bitmap; //symmetryOpsの時は代入します。
    symmetryOps_bitmap(size);
  }else{
    while(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap); //最も下位の１ビットを抽出
      backTrack2D(size,mask,row+1,(left|bit)<<1,down|bit,(right|bit)>>1);
    }
  }
}
//
void backTrack1D(int size,int mask,int row,int left,int down,int right){
  int bit;
  int bitmap=mask&~(left|down|right);   //BOUNDで対応済み
  if(row==size){
    aBoard[row]=bitmap; //symmetryOpsの時は代入します。
    symmetryOps_bitmap(size);
  }else{
    while(bitmap){
      bitmap^=aBoard[row]=bit=(-bitmap&bitmap); //最も下位の１ビットを抽出
      backTrack1D(size,mask,row+1,(left|bit)<<1,down|bit,(right|bit)>>1);
    }
  }
}
//
//CPUR 再帰版 ロジックメソッド
void NQueenDR(int size,int mask){
  int bit;
  TOPBIT=1<<(size-1);
  aBoard[0]=1;
  for(BOUND1=2;BOUND1<size-1;BOUND1++){
    aBoard[1]=bit=(1<<BOUND1);
    backTrack1D(size,mask,2,(2|bit)<<1,(1|bit),(bit>>1));
  }
  SIDEMASK=LASTMASK=(TOPBIT|1);
  ENDBIT=(TOPBIT>>1);
  for(BOUND1=1,BOUND2=size-2;BOUND1<BOUND2;BOUND1++,BOUND2--){
    aBoard[0]=bit=(1<<BOUND1);
    backTrack2D(size,mask,1,bit<<1,bit,bit>>1);
    LASTMASK|=LASTMASK>>1|LASTMASK<<1;
    ENDBIT>>=1;
  }
}
//
//メインメソッド
int main(int argc,char** argv) {
  bool cpu=false,cpur=false,gpu=false,sgpu=false;
  int argstart=1,steps=24576;
  /** パラメータの処理 */
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='r'||argv[1][1]=='R'){cpur=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else if(argv[1][1]=='s'||argv[1][1]=='S'){sgpu=true;}
    else
      cpur=true;
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g|-r|-s]\n",argv[0]);
    printf("  -c: CPU only\n");
    printf("  -r: CPUR only\n");
    printf("  -g: GPU only\n");
    printf("  -s: SGPU only\n");
    printf("Default to 8 queen\n");
  }
  /** 出力と実行 */
  if(cpu){
    printf("\n\n１０．CPU 非再帰 クイーンの位置による分岐BOUND1,2\n");
  }else if(cpur){
    printf("\n\n１０．CPUR 再帰 クイーンの位置による分岐BOUND1,2\n");
  }else if(gpu){
    printf("\n\n１０．GPU 非再帰 クイーンの位置による分岐BOUND1,2\n");
  }else if(sgpu){
    printf("\n\n１０．SGPU 非再帰 クイーンの位置による分岐BOUND1,2\n");
  }
  if(cpu||cpur){
    printf("%s\n"," N:        Total       Unique        hh:mm:ss.ms");
    clock_t st;           //速度計測用
    char t[20];           //hh:mm:ss.msを格納
    int min=4; int targetN=17;
    int mask;
    for(int i=min;i<=targetN;i++){
      //TOTAL=0; UNIQUE=0;
      COUNT2=COUNT4=COUNT8=0;
      mask=(1<<i)-1;
      st=clock();
      //初期化は不要です
      //非再帰は-1で初期化
      // for(int j=0;j<=targetN;j++){ aBoard[j]=-1; }
      //
      //CPUR
      if(cpur){ 
        NQueenR(i,mask); 
        //printf("通常版\n");
        //NQueenDR(i,mask);//通常版
      }
      //CPU
      if(cpu){ 
        NQueen(i,mask); 
        //printf("通常版\n");
        //NQueenD(i,mask);//通常版
      }
      //
      TimeFormat(clock()-st,t); 
      printf("%2d:%13ld%16ld%s\n",i,getTotal(),getUnique(),t);
    }
  }
  if(gpu||sgpu){
    if(!InitCUDA()){return 0;}
    int min=4;int targetN=18;int mask;
    struct timeval t0;struct timeval t1;int ss;int ms;int dd;
    printf("%s\n"," N:        Total      Unique      dd:hh:mm:ss.ms");
    long TOTAL,UNIQUE;
    long results[2];//結果格納用
    for(int i=min;i<=targetN;i++){
      gettimeofday(&t0,NULL);   // 計測開始
      if(gpu){
        mask=((1<<i)-1);
        solve_nqueen_cuda(i,mask,results,steps);
        TOTAL=results[0];
        UNIQUE=results[1];
      }else if(sgpu){
        TOTAL=sgpu_solve_nqueen_cuda(i,steps);
      }
      gettimeofday(&t1,NULL);   // 計測終了
      if (t1.tv_usec<t0.tv_usec) {
        dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      } else {
        dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%13ld%16ld%4.2d:%02d:%02d:%02d.%02d\n", i,TOTAL,UNIQUE,dd,hh,mm,ss,ms);
    }
  }
  return 0;
}
