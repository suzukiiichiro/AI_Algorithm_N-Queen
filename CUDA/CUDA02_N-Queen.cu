/**
 CUDAで学ぶアルゴリズムとデータ構造
 ステップバイステップでＮ−クイーン問題を最適化
 一般社団法人  共同通信社  情報技術局  鈴木  維一郎(suzuki.iichiro@kyodonews.jp)

 コンパイル
 $ nvcc CUDA02_N-Queen.cu -o CUDA02_N-Queen

 実行
 $ ./CUDA02_N-Queen (-c|-r|-g)
                    -c:cpu -r cpu再帰 -g GPU

 ２．配置フラグ（制約テスト高速化）

   パターンを生成し終わってからチェックを行うのではなく、途中で制約を満たさな
   い事が明らかな場合は、それ以降のパターン生成を行わない。
  「手を進められるだけ進めて、それ以上は無理（それ以上進めても解はない）という
  事がわかると一手だけ戻ってやり直す」という考え方で全ての手を調べる方法。
  (※)各行列に一個の王妃配置する組み合わせを再帰的に列挙分枝走査を行っても、組
  み合わせを列挙するだけであって、8王妃問題を解いているわけではありません。


  N-Queen の データ配列について
  =============================

  総当たり
  結局全部のケースをやってみる（完全解）

  バックトラック
  とりあえずやってみる。ダメなら戻って別の道を探る


  N-Queen: クイーンの効き筋
  =========================
  クイーンの位置から、縦、横、斜めが効き筋となります。

  　　       column(列)
  row(行)_0___1___2___3___4_
       0|-*-|---|---|-*-|---|
        +-------------------+
       1|---|-*-|---|-*-|---|
        +-------------------+ 
       2|---|---|-*-|-*-|-*-| 
        +-------------------+ 
       3|-*-|-*-|-*-|-Q-|-*-|
        +-------------------+
       4|---|---|-*-|-*-|-*-|
        +-------------------+


  N-Queen: 盤面上で互いのクイーンが効き筋にならないように配置
  ===========================================================

        完成図は以下の通りです。

  　　       column(列)
  row(行)_0___1___2___3___4_
       0|-Q-|---|---|---|---|
        +-------------------+
       1|---|---|---|-Q-|---|
        +-------------------+ 
       2|---|-Q-|---|---|---| 
        +-------------------+ 
       3|---|---|---|---|-Q-|
        +-------------------+
       4|---|---|-Q-|---|---|
        +-------------------+


  効き筋の表現
  ============

  クイーンの位置から下側を走査対象とします。

  　すでに効き筋：FALSE(盤面ではF）
  　配置可能    ：TRUE

  　　       column(列) 
  row(行)_0___1___2___3___4_
       0|---|---|---|---|---| 
        +-------------------+
       1|---|---|---|---|---|
        +-------------------+ 
       2|---|-Q-|---|---|---| 
        +-------------------+ 
       3|-F-|-F-|-F-|---|---|
        +-------------------+
       4|---|-F-|---|-F-|---|
        +-------------------+
                      

  効き筋を三つの配列で表現
  ========================

  ■ 基本：aBoard[row]=col
           aBoard[2  ]=1

  　　       column(列)
  row(行)_0___1___2___3___4_
       0|---|---|---|---|---|
        +-------------------+
       1|---|---|---|---|---|
        +-------------------+ 
       2|---|-Q-|---|---|---| aBoard[2]=1 に配置
        +-------------------+
       3|---|---|---|---|---|
        +-------------------+
       4|---|---|---|---|---|
        +-------------------+


  ■配列1：down[row]

  そのrow(行)にQueenがいる場合はFALSE
                      いない場合はTRUE

  　　       column(列)
  row(行)_0___1___2___3___4_
       0|---|---|---|---|---|
        +-------------------+
       1|---|---|---|---|---|
        +-------------------+ 
       2|---|-Q-|---|---|---| 
        +-------------------+
       3|---|-F-|---|---|---|
        +-------------------+
       4|---|-F-|---|---|---|
        +-------------------+
             down[col(1)]==false (すでに効き筋）


  ■配列２：right[col-row+N-1]
                    right[col-row+N-1]==F
                        Qの場所：col(1)-row(2)+(4-1)=2なので
                        col-row+N-1が２のところがＦとなる 
  　　       column(列)
  row(行)_0___1___2___3___4_
       0|---|---|---|---|---|
        +-------------------+
       1|---|---|---|---|---|
        +-------------------+ 
       2|---|-Q-|---|---|---| 
        +-------------------+ 
       3|---|---|-F-|---|---|
        +-------------------+
       4|---|---|---|-F-|---|
        +-------------------+
                      right[col-row+(N-1)]==false(すでに効き筋）


  ■配列3：left[col+row]
                      left[col+row]==F 
                          Qの場所：col(1)+row(2)=3なので
                          col+rowが3になるところがFとなる。

  　　       column(列) 
  row(行)_0___1___2___3___4_
       0|---|---|---|---|---|
        +-------------------+
       1|---|---|---|---|---|
        +-------------------+ 
       2|---|-Q-|---|---|---| 
        +-------------------+ 
       3|-F-|---|---|---|---|
        +-------------------+
       4|---|---|---|---|---|
        +-------------------+
      left[col+row]


  ステップ１
  ==========
  row=0, col=0 にクイーンを配置してみます。

  aBoard[row]=col
     ↓
  aBoard[0]=0;

  　　       column(列) 
  row(行)_0___1___2___3___4_
   ->  0|-Q-|---|---|---|---| aBoard[row]=col
        +-------------------+ aBoard[0  ]=0  
       1|---|---|---|---|---|
        +-------------------+ 
       2|---|---|---|---|---| 
        +-------------------+ 
       3|---|---|---|---|---|
        +-------------------+
       4|---|---|---|---|---|
        +-------------------+


  考え方：２
  ==========
  効き筋を埋めます

  　　       column(列) 
  row(行)_0___1___2___3___4_
   ->  0|-Q-|---|---|---|---| 
        +-------------------+ 
       1|-F-|-F-|---|---|---|
        +-------------------+ left はありません
       2|-F-|---|-F-|---|---| 
        +-------------------+ 
       3|-F-|---|---|-F-|---|
        +-------------------+
       4|-F-|---|---|---|-F-|
        +-------------------+
        down[col]      right[col-row+(N-1)]


  考え方：３
  ==========
  rowが一つ下に降りて０から１となります。
  次の候補は以下のＡ，Ｂ，Ｃとなります

  　　       column(列) 
  row(行)_0___1___2___3___4_
       0|-Q-|---|---|---|---| 
        +-------------------+ 
   ->  1|-F-|-F-|-A-|-B-|-C-|
        +-------------------+ 
       2|-F-|---|-F-|---|---| 
        +-------------------+ 
       3|-F-|---|---|-F-|---|
        +-------------------+
       4|-F-|---|---|---|-F-|
        +-------------------+

  考え方：４
  ==========
  Ａにおいてみます。
  効き筋は以下の通りです。

  　　       column(列) 
  row(行)_0___1___2___3___4_
       0|-Q-|---|---|---|---| 
        +-------------------+ 
   ->  1|-F-|-F-|-Q-|---|---|
        +-------------------+ 
       2|-F-|-F-|-F-|-F-|---| 
        +-------------------+ 
       3|-F-|---|-F-|-F-|-F-| right[col-row+(N-q)]
        +-------------------+
       4|-F-|---|-F-|---|-F-|
        +-------------------+
  left[col+row]  down[col]


  考え方：５
  ==========
  rowが一つ下に降りて１から２となります。
  次の候補はＡとなります

  　　       column(列) 
  row(行)_0___1___2___3___4_
       0|-Q-|---|---|---|---| 
        +-------------------+ 
       1|-F-|-F-|-Q-|---|---|
        +-------------------+ 
   ->  2|-F-|-F-|-F-|-F-|-A-| 
        +-------------------+ 
       3|-F-|---|-F-|-F-|-F-| 
        +-------------------+
       4|-F-|---|-F-|---|-F-|
        +-------------------+

  考え方：６
  ==========
  効き筋は以下の通りです。
  特に加わるところはありません。

  　　       column(列) 
  row(行)_0___1___2___3___4_
       0|-Q-|---|---|---|---| 
        +-------------------+ 
       1|-F-|-F-|-Q-|---|---|
        +-------------------+ 
   ->  2|-F-|-F-|-F-|-F-|-Q-| 
        +-------------------+ 
       3|-F-|---|-F-|-F-|-F-| 
        +-------------------+
       4|-F-|---|-F-|---|-F-|
        +-------------------+

  考え方：７
  ==========
  rowが一つ下に降りて２から３となります。
  次の候補はＡとなります

  　　       column(列) 
  row(行)_0___1___2___3___4_
       0|-Q-|---|---|---|---| 
        +-------------------+ 
       1|-F-|-F-|-Q-|---|---|
        +-------------------+ 
       2|-F-|-F-|-F-|-F-|-Q-| 
        +-------------------+ 
   ->  3|-F-|-A-|-F-|-F-|-F-| 
        +-------------------+
       4|-F-|---|-F-|---|-F-|
        +-------------------+


  考え方：８
  ==========
  効き筋は以下の通りです。

  　　       column(列) 
  row(行)_0___1___2___3___4_
       0|-Q-|---|---|---|---| 
        +-------------------+ 
       1|-F-|-F-|-Q-|---|---|
        +-------------------+ 
       2|-F-|-F-|-F-|-F-|-Q-| 
        +-------------------+ 
   ->  3|-F-|-Q-|-F-|-F-|-F-| 
        +-------------------+
       4|-F-|-F-|-F-|---|-F-|
        +-------------------+


  考え方：９
  ==========
  今回は、うまくいっていますが、
  次の候補がなければ、キャンセルして、
  前のコマを次の候補にコマを移動し、
  処理を継続します。


  考え方：１０
  =========-=

  rowが一つ下に降りて３から４となります。
  候補はのこり１箇所しかありません。

  　　       column(列) 
  row(行)_0___1___2___3___4_
       0|-Q-|---|---|---|---| 
        +-------------------+ 
       1|-F-|-F-|-Q-|---|---|
        +-------------------+ 
       2|-F-|-F-|-F-|-F-|-Q-| 
        +-------------------+ 
       3|-F-|-Q-|-F-|-F-|-F-| 
        +-------------------+
   ->  4|-F-|-F-|-F-|-A-|-F-|
        +-------------------+



  考え方：１１
  ==========
  最後のクイーンをおきます
  columnの最終列は効き筋を確認する必要はありませんね。

  　　       column(列) 
  row(行)_0___1___2___3___4_
       0|-Q-|---|---|---|---| 
        +-------------------+ 
       1|-F-|-F-|-Q-|---|---|
        +-------------------+ 
       2|-F-|-F-|-F-|-F-|-Q-| 
        +-------------------+ 
       3|-F-|-Q-|-F-|-F-|-F-| 
        +-------------------+
   ->  4|-F-|-F-|-F-|-Q-|-F-|
        +-------------------+

  考え方：１２
  ==========
  rowの脇にcolの位置を示します。

  　　       column(列) 
  row(行)_0___1___2___3___4_
       0|-Q-|---|---|---|---|  [0]
        +-------------------+ 
       1|-F-|-F-|-Q-|---|---|  [2]
        +-------------------+ 
       2|-F-|-F-|-F-|-F-|-Q-|  [4]
        +-------------------+ 
       3|-F-|-Q-|-F-|-F-|-F-|  [1]
        +-------------------+
   ->  4|-F-|-F-|-F-|-Q-|-F-|  [3]
        +-------------------+


  考え方：１３
  ==========

  ボード配列は以下のように表します。
  aBoard[]={0,2,4,1,3]

  出力：
    1: 0 0 0 1
    2: 0 0 0 2
    3: 0 0 0 4
    :
    :



 実行結果
$ nvcc CUDA02_N-Queen.cu  && ./a.out -r
 ２．CPUR 再帰 配置フラグ（制約テスト高速化）
 :
 :
111: 4 2 1 0 3 
112: 4 2 1 3 0 
113: 4 2 3 0 1 
114: 4 2 3 1 0 
115: 4 3 0 1 2 
116: 4 3 0 2 1 
117: 4 3 1 0 2 
118: 4 3 1 2 0 
119: 4 3 2 0 1 
120: 4 3 2 1 0 

$ nvcc CUDA02_N-Queen.cu  && ./a.out -c
 ２．CPU 非再帰 配置フラグ（制約テスト高速化）
 :
 :
111: 4 2 1 0 3 
112: 4 2 1 3 0 
113: 4 2 3 0 1 
114: 4 2 3 1 0 
115: 4 3 0 1 2 
116: 4 3 0 2 1 
117: 4 3 1 0 2 
118: 4 3 1 2 0 
119: 4 3 2 0 1 
120: 4 3 2 1 0 

$ nvcc CUDA02_N-Queen.cu  && ./a.out -g
２. GPU 非再帰 配置フラグ（制約テスト高速化）


*/

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>


#define THREAD_NUM		96
#define MAX 27
//変数宣言
long Total=0 ;      //GPU
long Unique=0;			//GPU
int COUNT=0;     		//カウント用
int aBoard[MAX]; 		//版の配列
int down[2*MAX-1]; 	//down:flagA 縦 配置フラグ　
//関数宣言
void print(int size);
void NQueen(int row,int size);
void NQueenR(int row,int size);
//
__global__ void solve_nqueen_cuda_kernel_bt_bm(
  int n,int mark,
  unsigned int* totalDown,unsigned int* totalLeft,unsigned int* totalRight,
  unsigned int* results,int totalCond){
  const int tid=threadIdx.x,bid=blockIdx.x,idx=bid*blockDim.x+tid;
  __shared__ unsigned int down[THREAD_NUM][10],left[THREAD_NUM][10],right[THREAD_NUM][10],
                          bitmap[THREAD_NUM][10],sum[THREAD_NUM];
  const unsigned int mask=(1<<n)-1;int total=0,i=0;unsigned int bit;
  if(idx<totalCond){
    down[tid][i]=totalDown[idx];
    left[tid][i]=totalLeft[idx];
    right[tid][i]=totalRight[idx];
    bitmap[tid][i]=down[tid][i]|left[tid][i]|right[tid][i];
    while(i>=0){
      if((bitmap[tid][i]&mask)==mask){i--;}
      else{
        bit=(bitmap[tid][i]+1)&~bitmap[tid][i];
        bitmap[tid][i]|=bit;
        if((bit&mask)!=0){
          if(i+1==mark){total++;i--;}
          else{
            down[tid][i+1]=down[tid][i]|bit;
            left[tid][i+1]=(left[tid][i]|bit)<<1;
            right[tid][i+1]=(right[tid][i]|bit)>>1;
            bitmap[tid][i+1]=(down[tid][i+1]|left[tid][i+1]|right[tid][i+1]);
            i++;
          }
        }else{i--;}
      }
    }
    sum[tid]=total;
  }else{sum[tid]=0;} 
  __syncthreads();if(tid<64&&tid+64<THREAD_NUM){sum[tid]+=sum[tid+64];} 
  __syncthreads();if(tid<32){sum[tid]+=sum[tid+32];} 
  __syncthreads();if(tid<16){sum[tid]+=sum[tid+16];} 
  __syncthreads();if(tid<8){sum[tid]+=sum[tid+8];} 
  __syncthreads();if(tid<4){sum[tid]+=sum[tid+4];} 
  __syncthreads();if(tid<2){sum[tid]+=sum[tid+2];} 
  __syncthreads();if(tid<1){sum[tid]+=sum[tid+1];} 
  __syncthreads();if(tid==0){results[bid]=sum[0];}
}
//
long long solve_nqueen_cuda(int n,int steps) {
  unsigned int down[32];unsigned int left[32];unsigned int right[32];
  unsigned int m[32];unsigned int bit;
  if(n<=0||n>32){return 0;}
  unsigned int* totalDown=new unsigned int[steps];
  unsigned int* totalLeft=new unsigned int[steps];
  unsigned int* totalRight=new unsigned int[steps];
  unsigned int* results=new unsigned int[steps];
  unsigned int* downCuda;unsigned int* leftCuda;unsigned int* rightCuda;
  unsigned int* resultsCuda;
  hipMalloc((void**) &downCuda,sizeof(int)*steps);
  hipMalloc((void**) &leftCuda,sizeof(int)*steps);
  hipMalloc((void**) &rightCuda,sizeof(int)*steps);
  hipMalloc((void**) &resultsCuda,sizeof(int)*steps/THREAD_NUM);
  const unsigned int mask=(1<<n)-1;
  const unsigned int mark=n>11?n-10:2;
  long long total=0;int totalCond=0;
  int i=0,j;down[0]=0;left[0]=0;right[0]=0;m[0]=0;bool computed=false;
  for(j=0;j<n/2;j++){
    bit=(1<<j);m[0]|=bit;
    down[1]=bit;left[1]=bit<<1;right[1]=bit>>1;
    m[1]=(down[1]|left[1]|right[1]);
    i=1;
    while(i>0){
      if((m[i]&mask)==mask){i--;}
      else{
        bit=(m[i]+1)&~m[i];m[i]|=bit;
        if((bit&mask)!=0){
          down[i+1]=down[i]|bit;left[i+1]=(left[i]|bit)<<1;right[i+1]=(right[i]|bit)>>1;
          m[i+1]=(down[i+1]|left[i+1]|right[i+1]);
          i++;
          if(i==mark){
            totalDown[totalCond]=down[i];totalLeft[totalCond]=left[i];totalRight[totalCond]=right[i];
            totalCond++;
            if(totalCond==steps){
              if(computed){
                hipMemcpy(results,resultsCuda,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int j=0;j<steps/THREAD_NUM;j++){total+=results[j];}
                computed=false;
              }
              hipMemcpy(downCuda,totalDown,sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              solve_nqueen_cuda_kernel_bt_bm<<<steps/THREAD_NUM,THREAD_NUM>>>(n,n-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              computed=true;totalCond=0;
            }
            i--;
          }
        }else{i --;}
      }
    }
  }
  if(computed){
    hipMemcpy(results,resultsCuda,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int j=0;j<steps/THREAD_NUM;j++){total+=results[j];}
    computed=false;
  }
  hipMemcpy(downCuda,totalDown,sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(leftCuda,totalLeft,sizeof(int)*totalCond,hipMemcpyHostToDevice);
  hipMemcpy(rightCuda,totalRight,sizeof(int)*totalCond,hipMemcpyHostToDevice);
  /** backTrack+bitmap*/
  solve_nqueen_cuda_kernel_bt_bm<<<steps/THREAD_NUM,THREAD_NUM>>>(n,n-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
  hipMemcpy(results,resultsCuda,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
  for(int j=0;j<steps/THREAD_NUM;j++){total+=results[j];}	
  total*=2;
  if(n%2==1){
    computed=false;totalCond=0;bit=(1<<(n-1)/2);m[0]|=bit;
    down[1]=bit;left[1]=bit<<1;right[1]=bit>>1;
    m[1]=(down[1]|left[1]|right[1]);
    i=1;
    while(i>0){
      if((m[i]&mask)==mask){i--;}
      else{
        bit=(m[i]+1)&~m[i];m[i]|=bit;
        if((bit&mask)!=0){
          down[i+1]=down[i]|bit;left[i+1]=(left[i]|bit)<<1;right[i+1]=(right[i]|bit)>>1;
          m[i+1]=(down[i+1]|left[i+1]|right[i+1]);
          i++;
          if(i==mark){
            totalDown[totalCond]=down[i];totalLeft[totalCond]=left[i];totalRight[totalCond]=right[i];
            totalCond++;
            if(totalCond==steps){
              if(computed){
                hipMemcpy(results,resultsCuda,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
                for(int j=0;j<steps/THREAD_NUM;j++){total+=results[j];}
                computed=false;
              }
              hipMemcpy(downCuda,totalDown,sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(leftCuda,totalLeft,sizeof(int)*totalCond,hipMemcpyHostToDevice);
              hipMemcpy(rightCuda,totalRight,sizeof(int)*totalCond,hipMemcpyHostToDevice);
              /** backTrack+bitmap*/
              solve_nqueen_cuda_kernel_bt_bm<<<steps/THREAD_NUM,THREAD_NUM>>>(n,n-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
              computed=true;totalCond=0;
            }
            i--;
          }
        }else{i --;}
      }
    }
    if(computed){
      hipMemcpy(results,resultsCuda,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
      for(int j=0;j<steps/THREAD_NUM;j++){total+=results[j];}
      computed=false;
    }
    hipMemcpy(downCuda,totalDown,sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(leftCuda,totalLeft,sizeof(int)*totalCond,hipMemcpyHostToDevice);
    hipMemcpy(rightCuda,totalRight,sizeof(int)*totalCond,hipMemcpyHostToDevice);
    /** backTrack+bitmap*/
    solve_nqueen_cuda_kernel_bt_bm<<<steps/THREAD_NUM,THREAD_NUM>>>(n,n-mark,downCuda,leftCuda,rightCuda,resultsCuda,totalCond);
    hipMemcpy(results,resultsCuda,sizeof(int)*steps/THREAD_NUM,hipMemcpyDeviceToHost);
    for(int j=0;j<steps/THREAD_NUM;j++){total+=results[j];}
  }
  hipFree(downCuda);hipFree(leftCuda);hipFree(rightCuda);hipFree(resultsCuda);
  delete[] totalDown;delete[] totalLeft;delete[] totalRight;delete[] results;
  return total;
}
/** CUDA 初期化 **/
bool InitCUDA(){
  int count;
  hipGetDeviceCount(&count);
  if(count==0){fprintf(stderr,"There is no device.\n");return false;}
  int i;
  for(i=0;i<count;i++){
    hipDeviceProp_t prop;
    if(hipGetDeviceProperties(&prop,i)==hipSuccess){if(prop.major>=1){break;} }
  }
  if(i==count){fprintf(stderr,"There is no device supporting CUDA 1.x.\n");return false;}
  hipSetDevice(i);
  return true;
}
//出力用のメソッド
void print(int size){
	printf("%d: ",++COUNT);
	for(int j=0;j<size;j++){
		printf("%d ",aBoard[j]);
	}
	printf("\n");
}
//CPU 非再帰 ロジックメソッド
void NQueen(int row,int size){
  bool matched;
  while(row>=0){
    matched=false;
    for(int col=aBoard[row]+1;col<size;col++){
      if(down[col]==0){      //downは効き筋ではない
        if(aBoard[row]!=-1){ //Qは配置済み
          down[aBoard[row]]=0;//downの効き筋を外す
        }
        aBoard[row]=col;     //Qを配置
        down[col]=1;         //downは効き筋である
        matched=true;
        break;
      }
    }
    if(matched){
      row++;
      if(row==size){
        print(size);
        row--;
      }
    }else{                   //置けるところがない
      if(aBoard[row]!=-1){
        int col=aBoard[row]; /** colの代用 */
        down[col]=0;         //downの効き筋を解除
        aBoard[row]=-1;      //空き地に戻す
      }
      row--;
    }
  }
}
//CPUR 再帰 ロジックメソッド
void NQueenR(int row,int size){
  if(row==size){
    print(size);
  }else{
    for(int col=aBoard[row]+1;col<size;col++){
      aBoard[row]=col;  //Qを配置
      if(down[col]==0){
        down[col]=1;
        NQueenR(row+1,size);
        down[col]=0;
      }
      aBoard[row]=-1;   //空き地に戻す
    }
  }
}
//メインメソッド
int main(int argc,char** argv) {
	int size=5;
  bool cpu=false,cpur=false,gpu=false;
  int argstart=1,steps=24576;
  /** パラメータの処理 */
  if(argc>=2&&argv[1][0]=='-'){
    if(argv[1][1]=='c'||argv[1][1]=='C'){cpu=true;}
    else if(argv[1][1]=='r'||argv[1][1]=='R'){cpur=true;}
    else if(argv[1][1]=='g'||argv[1][1]=='G'){gpu=true;}
    else
      cpur=true;
    argstart=2;
  }
  if(argc<argstart){
    printf("Usage: %s [-c|-g|-r] n steps\n",argv[0]);
    printf("  -c: CPU only\n");
    printf("  -r: CPUR only\n");
    printf("  -g: GPU only\n");
    printf("Default CPUR to 8 queen\n");
  }
  /** 出力と実行 */
	//aBoard配列を-1で初期化
  for(int i=0;i<size;i++){ aBoard[i]=-1; }
  /** CPU */
  if(cpu){
    printf("\n\n２．CPU 非再帰 配置フラグ（制約テスト高速化）\n");
    NQueen(0,size);
  }
  /** CPUR */
  if(cpur){
    printf("\n\n２．CPUR 再帰 配置フラグ（制約テスト高速化）\n");
    NQueenR(0,size);//ロジックメソッドを0を渡して呼び出し
  }
  /** GPU */
  if(gpu){
    if(!InitCUDA()){return 0;}
    int min=4;int targetN=18;
    struct timeval t0;struct timeval t1;int ss;int ms;int dd;
    printf("\n\n2. GPU 非再帰 配置フラグ（制約テスト高速化）\n");
    printf("%s\n"," N:          Total        Unique                 dd:hh:mm:ss.ms");
    for(int i=min;i<=targetN;i++){
      gettimeofday(&t0,NULL);   // 計測開始
      Total=solve_nqueen_cuda(i,steps);
      gettimeofday(&t1,NULL);   // 計測終了
      if (t1.tv_usec<t0.tv_usec) {
        dd=(int)(t1.tv_sec-t0.tv_sec-1)/86400;
        ss=(t1.tv_sec-t0.tv_sec-1)%86400;
        ms=(1000000+t1.tv_usec-t0.tv_usec+500)/10000;
      } else {
        dd=(int)(t1.tv_sec-t0.tv_sec)/86400;
        ss=(t1.tv_sec-t0.tv_sec)%86400;
        ms=(t1.tv_usec-t0.tv_usec+500)/10000;
      }
      int hh=ss/3600;
      int mm=(ss-hh*3600)/60;
      ss%=60;
      printf("%2d:%18ld%18ld%12.2d:%02d:%02d:%02d.%02d\n", i,Total,Unique,dd,hh,mm,ss,ms);
    }
  }
  return 0;
}
